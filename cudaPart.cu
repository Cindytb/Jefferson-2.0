#include "hip/hip_runtime.h"
#include "cudaPart.cuh"

void cudaFFT(int argc, char **argv, Data *p){
	findCudaDevice(argc, (const char **)argv);
	int SIGNAL_SIZE = 0, FILTER_KERNEL_SIZE = 0;
	
	/*SNDFILE SETUP & READING FILES*/
	SF_INFO isfinfo, rsfinfo;
	SNDFILE *isndfile, *rsndfile;
	char *input = "Vltava.wav";
	char *reverb = "s1_r1_b_441_mono.wav";
	/*if (argc == 1) {
		input = argv[1];
	}
	if (argc == 2) {
		input = argv[1];
		reverb = argv[2];
	}*/

	/*Buffers for wave files*/
	float *ibuf, *rbuf;
	memset(&isfinfo, 0, sizeof(isfinfo));
	memset(&rsfinfo, 0, sizeof(rsfinfo));

	/*Open input file*/
	isfinfo.format = 0;
	isndfile = sf_open(input, SFM_READ, &isfinfo);
	if (isndfile == NULL) {
		fprintf(stderr, "ERROR. Cannot open %s\n", input);
		exit(1);
	}
	if (isfinfo.channels != 1) {
		printf("ERROR: Only mono sources accepted");
		exit(2);
	}
	if (isfinfo.samplerate != 44100) {
		printf("ERROR: Only 44.1k SR accepted");
		exit(3);
	}
	/*Open reverb file*/
	rsfinfo.format = 0;
	rsndfile = sf_open(reverb, SFM_READ, &rsfinfo);
	if (rsndfile == NULL) {
		printf("ERROR. Cannot open %s\n", reverb);
		exit(1);
	}
	if (rsfinfo.channels != 1) {
		printf("ERROR: Only mono sources accepted");
		exit(2);
	}
	if (rsfinfo.samplerate != 44100) {
		printf("ERROR: Only 44.1k SR accepted");
		exit(3);
	}
	

	/*Setup buffer sizes*/
	SIGNAL_SIZE = (int)(isfinfo.channels * isfinfo.frames);
	FILTER_KERNEL_SIZE = (int)(rsfinfo.channels * rsfinfo.frames);

	/*Allocate buffers*/
	ibuf = (float*)malloc(sizeof(float) * SIGNAL_SIZE);
	rbuf = (float*)malloc(sizeof(float) * FILTER_KERNEL_SIZE);

	/*Read buffers*/
	sf_read_float(isndfile, ibuf, SIGNAL_SIZE);
	sf_read_float(rsndfile, rbuf, FILTER_KERNEL_SIZE);
	printf("First data: %f\n", ibuf[0] * rbuf[0]);

	/*Pad signal and filter kernel to same length*/
	float *h_padded_signal;
	float *h_padded_filter_kernel;
	//new_size = SIGNAL_SIZE + (FILTER_KERNEL_SIZE) % 2
	int new_size = PadData(ibuf, &h_padded_signal, SIGNAL_SIZE,
		rbuf, &h_padded_filter_kernel, FILTER_KERNEL_SIZE);
	int mem_size = sizeof(float) * new_size;


	/*MOVING SIGNAL TO GPU*/
	// Allocate device memory for signal
	float *d_signal;
	checkCudaErrors(hipMalloc((void **)&d_signal, mem_size));

	// Copy signal from host to device
	checkCudaErrors(hipMemcpy(d_signal, h_padded_signal, mem_size,
		hipMemcpyHostToDevice));

	/*MOVING IMPULSE TO GPU*/
	// Allocate device memory for filter kernel
	float *d_filter_kernel;
	checkCudaErrors(hipMalloc((void **)&d_filter_kernel, mem_size));
	// Copy host memory to device
	checkCudaErrors(hipMemcpy(d_filter_kernel, h_padded_filter_kernel, mem_size,
		hipMemcpyHostToDevice));

	/*FIND RMS OF ORIGINAL SIGNAL*/
	/*Convert raw float pointer into a thrust device pointer*/
	thrust::device_ptr<float> thrust_d_signal(d_signal);
	/*Declare thrust operators*/
	square<float> unary_op;
	thrust::plus<float> binary_op;
	/*Perform thrust reduction to find rms*/
	float rms = std::sqrt(thrust::transform_reduce(thrust_d_signal, thrust_d_signal + new_size, unary_op, 0.0f, binary_op) / new_size);
	
	///////////////////////////////////////////////////////////////////////////////
	/*GPU PROCESSING*/
	///////////////////////////////////////////////////////////////////////////////
	
	// CUFFT plan simple API
	hipfftHandle plan;
	checkCudaErrors(hipfftPlan1d(&plan, new_size, HIPFFT_R2C, 1));
	hipfftHandle outplan;
	checkCudaErrors(hipfftPlan1d(&outplan, new_size, HIPFFT_C2R, 1));

	/*Create complex arrays*/
	hipfftComplex *d_sig_complex;
	checkCudaErrors(hipMalloc(&d_sig_complex, new_size * sizeof(hipfftComplex)));
	hipfftComplex *d_filter_complex;
	checkCudaErrors(hipMalloc(&d_filter_complex, new_size * sizeof(hipfftComplex)));

	/*FFT*/
	printf("Transforming signal hipfftExecR2C\n");
	checkCudaErrors(hipfftExecR2C(plan, (hipfftReal *)d_signal, d_sig_complex));
	checkCudaErrors(hipfftExecR2C(plan, (hipfftReal *)d_filter_kernel, d_filter_complex));

	/*CONVOLUTION*/
	// Multiply the coefficients together and normalize the result
	printf("Launching ComplexPointwiseMulAndScale<<< >>>\n");
	int blockSize = 256;
	int numBlocks = (new_size + blockSize - 1) / blockSize;
	ComplexPointwiseMulAndScale << < numBlocks, blockSize >> > (d_sig_complex, d_filter_complex, new_size, 1.0f / new_size);
	// Check if kernel execution generated and error
	getLastCudaError("Kernel execution failed [ ComplexPointwiseMulAndScale ]");

	/*IFFT*/
	// Transform signal back
	printf("Transforming signal back hipfftExecC2R\n");
	checkCudaErrors(hipfftExecC2R(outplan, d_sig_complex, d_signal));

	if (hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "Cuda error: failed to synchronize\n");
	}

	/*Find RMS of resulting signal*/
	thrust::device_ptr<float> thrust_d_output_signal2(d_signal);
	float rms2 = std::sqrt(thrust::transform_reduce(thrust_d_signal, thrust_d_signal + new_size, unary_op, 0.0f, binary_op) / new_size);
	printf("RMS1: %f RMS2: %f\n", rms, rms2);

	/*Scale resulting signal according to input signal*/
	MyFloatScale << < numBlocks, blockSize >> > (d_signal, new_size, rms / rms2);
	
	/*MOVE BACK TO CPU & STORE IN STRUCT*/
	float *obuf = (float*)malloc(sizeof(float) * new_size);
	checkCudaErrors(hipMemcpy(obuf, d_signal, new_size * sizeof(float), hipMemcpyDeviceToHost));
	p->buf = obuf;
	p->length = new_size;


	////////////////////////////////////////////////////////////////////////////////
	///*NOTE: GPU Convolution was not fast enough because of the large overhead
	//of FFT and IFFT. Keeping the code here for future purposes*/
	//
	/*Convolved signal on device*/
	//p->dbuf = d_signal;
	////////////////////////////////////////////////////////////////////////////////

	/*Destroy CUFFT context*/
	checkCudaErrors(hipfftDestroy(plan));
	checkCudaErrors(hipfftDestroy(outplan));
	isfinfo.seekable = 1;

	/*Write sound file*/
	SndfileHandle file = SndfileHandle("output.wav", SFM_WRITE, isfinfo.format, isfinfo.channels, isfinfo.samplerate);
	file.write(obuf, new_size);


	/*Free memory*/
	sf_close(isndfile);
	sf_close(rsndfile);

	free(ibuf);
	free(rbuf);

	free(h_padded_signal);
	free(h_padded_filter_kernel);

	checkCudaErrors(hipFree(d_signal));
	checkCudaErrors(hipFree(d_filter_kernel));
	checkCudaErrors(hipFree(d_sig_complex));
	checkCudaErrors(hipFree(d_filter_complex));


}

// Pad data
int PadData(const float *signal, float **padded_signal, int signal_size,
	const float *filter_kernel, float **padded_filter_kernel, int filter_kernel_size)
{
	int minRadius = filter_kernel_size / 2;
	int maxRadius = filter_kernel_size - minRadius;
	int new_size = signal_size + maxRadius;

	// Pad signal
	float *new_data = (float *)malloc(sizeof(float) * new_size);
	memcpy(new_data + 0, signal, signal_size * sizeof(float));
	memset(new_data + signal_size, 0, (new_size - signal_size) * sizeof(float));
	*padded_signal = new_data;

	// Pad filter
	new_data = (float *)malloc(sizeof(float) * new_size);
	memcpy(new_data + 0, filter_kernel, filter_kernel_size * sizeof(float));
	memset(new_data + filter_kernel_size, 0, (new_size - filter_kernel_size) * sizeof(float));
	*padded_filter_kernel = new_data;
	return new_size;
}

// Complex scale
static __device__ __host__ inline Complex ComplexScale(Complex a, float s)
{
	Complex c;
	c.x = s * a.x;
	c.y = s * a.y;
	return c;
}
// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b)
{
	Complex c;
	c.x = a.x * b.x - a.y * b.y;
	c.y = a.x * b.y + a.y * b.x;
	return c;
}
// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulAndScale(Complex *a, const Complex *b, int size, float scale)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < size; i += numThreads)
	{
		a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);
	}
}
static __global__ void MyFloatScale(float *a, int size, float scale) {
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < size; i += numThreads)
	{
		a[i] = a[i] * scale;
	}
}
////////////////////////////////////////////////////////////////////////////////
///*NOTE: GPU Convolution was not fast enough because of the large overhead
//of FFT and IFFT. Keeping the code here for future purposes*/
//
//void __global__ padData(int size, float *padder) {
//	const int numThreads = blockDim.x * gridDim.x;
//	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
//
//	for (int i = threadID; i < size; i += numThreads)
//	{
//		padder[i] = 0.0f;
//	}
//}
//static __global__ void interleaveMe(float *output, float *input, int size) {
//	const int numThreads = blockDim.x * gridDim.x;
//	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
//
//	for (int i = threadID; i < size; i += numThreads)
//	{
//		output[i * 2 + 1] = input[i];
//		output[i * 2] = input[i];
//	}
//}
//
//__global__ void copyMe(int size, float *output, float *input) {
//	const int numThreads = blockDim.x * gridDim.x;
//	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
//
//	for (int i = threadID; i < size; i += numThreads)
//	{
//		output[i] = input[i];
//	}
//}
//
//void convolveMe(float *output, float *input, int x_len, float *p_hrtf, float gain, float *d_hrtf) {
//	int outputLength = x_len * 2;
//
//	int blockSize = 256;
//	int numBlocks = (x_len * 2 + blockSize - 1) / blockSize;
//	/*Interleave the input signal*/
//	float *d_interleaved;
//	checkCudaErrors(hipMalloc((void **)&d_interleaved, outputLength * sizeof(float)));
//	interleaveMe << < numBlocks, blockSize >> > (d_interleaved, input, x_len);
//	hipDeviceSynchronize();
//	
//
//	/*pad the HRTF signal*/
//	float *d_padded_hrtf;
//	checkCudaErrors(hipMalloc((void **)&d_padded_hrtf, outputLength * sizeof(float)));
//	padData << < numBlocks, blockSize >> > (x_len * 2 - HRTF_LEN, &d_padded_hrtf[HRTF_LEN]);
//	hipDeviceSynchronize();
//	copyMe << < numBlocks, blockSize >> > (HRTF_LEN, d_padded_hrtf, d_hrtf);
//	hipDeviceSynchronize();
//
//
//	/*CUFFT plan simple API*/
//	hipfftHandle plan;
//	checkCudaErrors(hipfftPlan1d(&plan, outputLength, HIPFFT_R2C, 1));
//	hipfftHandle outplan;
//	checkCudaErrors(hipfftPlan1d(&outplan, outputLength, HIPFFT_C2R, 1));
//
//	//Create complex arrays
//	hipfftComplex *d_sig_complex;
//	checkCudaErrors(hipMalloc(&d_sig_complex, outputLength * sizeof(hipfftComplex)));
//	hipfftComplex *d_filter_complex;
//	checkCudaErrors(hipMalloc(&d_filter_complex, outputLength * sizeof(hipfftComplex)));
//	
//	/*FFT*/
//	checkCudaErrors(hipfftExecR2C(plan, (hipfftReal *)d_interleaved, d_sig_complex));
//	checkCudaErrors(hipfftExecR2C(plan, (hipfftReal *)d_padded_hrtf, d_filter_complex));
//	
//	/*CONVOLUTION*/
//	//Multiply the coefficients together and normalize the result
//	ComplexPointwiseMulAndScale <<< numBlocks, blockSize >> > (d_sig_complex, d_filter_complex, outputLength, 1.0f / outputLength);
//	hipDeviceSynchronize();
//	//Check if kernel execution generated and error
//	getLastCudaError("Kernel execution failed [ ComplexPointwiseMulAndScale ]");
//	hipDeviceSynchronize();
//
//	/*IFFT*/
//	checkCudaErrors(hipfftExecC2R(outplan, d_sig_complex, d_interleaved));
//	if (hipDeviceSynchronize() != hipSuccess) {
//		fprintf(stderr, "Cuda error: failed to synchronize\n");
//	}
//	
//	/*Copy result into output*/
//	checkCudaErrors(hipMemcpy(output, d_interleaved, x_len * 2 * sizeof(float), hipMemcpyDeviceToHost));
//
//	//Destroy CUFFT context
//	checkCudaErrors(hipfftDestroy(plan));
//	checkCudaErrors(hipfftDestroy(outplan));
//
//	/*Free memory*/
//	hipFree(d_interleaved);
//	hipFree(d_padded_hrtf);
//	hipFree(d_sig_complex);
//	hipFree(d_filter_complex);
//}
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

