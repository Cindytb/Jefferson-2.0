#include "hip/hip_runtime.h"
#include "main.cuh"

int main(int argc, char *argv[]){
	if (argc < 1) {
		printf("Usage: %s input.wav reverb.wav", argv[0]);
		return 0;
	}
	/*Initialize data structure*/
	Data data, *p = &data;
	data.count = 0;
	data.length = 0;
	data.gain = 0.99074;
	data.hrtf_idx = -151;
	#if(DEBUGMODE != 1)
		/*Initialize & read files*/
		cudaFFT(argc, argv, p);
		SNDFILE *test;
		SF_INFO test1;
		test = sf_open("scrap.wav", SFM_READ, &test1);
	
	////////////////////////////////////////////////////////////////////////////////
	///*NOTE: GPU Convolution was not fast enough because of the large overhead
	//of FFT and IFFT. Keeping the code here for future purposes*/
	//
	//checkCudaErrors(hipMalloc((void**)&p->d_x, HRTF_LEN - 1 + FRAMES_PER_BUFFER));
	////////////////////////////////////////////////////////////////////////////////
	/*Open & read hrtf files*/
	if (read_hrtf_signals() != 0) {
		exit(EXIT_FAILURE);
	}
	p->hrtf_idx = 0;
	for (int i = 0; i < sizeof(p->x) / sizeof(*p->x); i++) {
		p->x[i] = 0.0;
	}
	p->osfinfo.channels = 2;
	p->osfinfo.samplerate = 44100;
	p->osfinfo.format = test1.format;
	p->sndfile = sf_open("ofile.wav", SFM_WRITE, &p->osfinfo);
	sf_close(test);
	#endif

	PaStream *stream = NULL;
	PaError err = NULL;
	

#if(DEBUGMODE != 1)
	PaStreamParameters outputParams;
	PaStreamParameters inputParams;
	
	/* Initializing PortAudio */
	err = Pa_Initialize();
	if (err != paNoError) {
		printf("PortAudio error: %s\n", Pa_GetErrorText(err));
		printf("\nExiting.\n");
		exit(1);
	}

	/* Input stream parameters */
	inputParams.device = Pa_GetDefaultInputDevice();
	inputParams.channelCount = 1;
	inputParams.sampleFormat = paFloat32;
	inputParams.suggestedLatency =
		Pa_GetDeviceInfo(inputParams.device)->defaultLowInputLatency;
	inputParams.hostApiSpecificStreamInfo = NULL;

	/* Ouput stream parameters */
	outputParams.device = Pa_GetDefaultOutputDevice();
	outputParams.channelCount = 2;
	outputParams.sampleFormat = paFloat32;
	outputParams.suggestedLatency =
		Pa_GetDeviceInfo(outputParams.device)->defaultLowOutputLatency;
	outputParams.hostApiSpecificStreamInfo = NULL;

	/* Open audio stream */
	err = Pa_OpenStream(&stream,
		&inputParams, /* no input */
		&outputParams,
		SAMPLE_RATE, FRAMES_PER_BUFFER,
		paNoFlag, /* flags */
		paCallback,
		&data);

	if (err != paNoError) {
		printf("PortAudio error: open stream: %s\n", Pa_GetErrorText(err));
		printf("\nExiting.\n");
		exit(1);
	}

	/* Start audio stream */
	err = Pa_StartStream(stream);
	if (err != paNoError) {
		printf("PortAudio error: start stream: %s\n", Pa_GetErrorText(err));
		printf("\nExiting.\n");
		exit(1);
	}

	printf("Starting playout\n");

#endif
	///////////////////////////////////////////////////////////////////////////////////////////////////
	/*MAIN FUNCTIONAL LOOP*/
	/*Here to debug without graphics*/
#if DEBUGMODE == 2
	char merp = getchar();
#else
	graphicsMain(argc, argv, p, err, stream);
#endif
	
	///////////////////////////////////////////////////////////////////////////////////////////////////
	
	/*THIS SECTION WILL NOT RUN IF GRAPHICS IS TURNED ON*/
	/*Placed here to properly close files when debugging without graphics*/
	
	/*Close output file*/
	sf_close(p->sndfile);

	/* Stop stream */
	err = Pa_StopStream(stream);
	if (err != paNoError) {
		printf("PortAudio error: stop stream: %s\n", Pa_GetErrorText(err));
		printf("\nExiting.\n");
		exit(1);
	}

	/* Close stream */
	err = Pa_CloseStream(stream);
	if (err != paNoError) {
		printf("PortAudio error: close stream: %s\n", Pa_GetErrorText(err));
		printf("\nExiting.\n");
		exit(1);
	}

	/* Terminate PortAudio */
	err = Pa_Terminate();
	if (err != paNoError) {
		printf("PortAudio error: terminate: %s\n", Pa_GetErrorText(err));
		printf("\nExiting.\n");
		exit(1);
	}

	return 0;
}

/* This routine will be called by the PortAudio engine when audio is needed.
* It may called at interrupt level on some machines so don't do anything
* in the routine that requires significant resources.
*/
static int paCallback(const void *inputBuffer, void *outputBuffer,
	unsigned long framesPerBuffer,
	const PaStreamCallbackTimeInfo* timeInfo,
	PaStreamCallbackFlags statusFlags,
	void *userData)
{
	/* Cast data passed through stream to our structure. */
	Data *p = (Data *)userData;
	float *output = (float *)outputBuffer;
	//float *input = (float *)inputBuffer; /* input not used in this code */
	float *px;
	int i;
	float *buf = (float*)malloc(sizeof(float) * 2 * framesPerBuffer - HRTF_LEN);

	/*CPU/RAM Copy data loop*/
	for (int i = 0; i < framesPerBuffer; i++) {
		p->x[HRTF_LEN - 1 + i] = p->buf[p->count];
		p->count++;
		if (p->count == p->length) {
			p->count = 0;
		}
	}
	/*convolve with HRTF on CPU*/
	convolve_hrtf(&p->x[HRTF_LEN], p->hrtf_idx, output, framesPerBuffer, p->gain);
	
	
	////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////
	/*NOTE: GPU Convolution was not fast enough because of the large overhead
	of FFT and IFFT. Keeping the code here for future purposes*/
	/*CUDA Copy*/
	//hipDeviceSynchronize();
	//int blockSize = 256;
	//int numBlocks = (framesPerBuffer + blockSize - 1) / blockSize;
	//if(p->count + framesPerBuffer <= p->length) {
	//	copyMe << < numBlocks, blockSize >> > (framesPerBuffer, p->d_x, &p->dbuf[p->count]);
	//	hipDeviceSynchronize();
	//	p->count += framesPerBuffer;
	//}
	//
	//else {
	//	int remainder = p->length - p->count - framesPerBuffer;
	//	copyMe << < numBlocks, blockSize >> > (p->length - p->count, p->d_x, &p->dbuf[p->count]);
	//	p->count = 0;
	//	copyMe << < numBlocks, blockSize >> > (remainder, p->d_x, &p->dbuf[p->count]);
	//	p->count += remainder;
	//}
	/*Convolve on GPU*/
	//GPUconvolve_hrtf(p->d_x, framesPerBuffer, p->hrtf_idx, output, framesPerBuffer, p->gain);
	////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////


	/* copy last HRTF_LEN-1 samples of x data to "history" part of x for use next time */
	px = p->x;
	for (i = 0; i<HRTF_LEN - 1; i++) {
		px[i] = px[framesPerBuffer + i];
	}
	//sf_writef_float(p->sndfile, output, framesPerBuffer);
	return 0;
}
