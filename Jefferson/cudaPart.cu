#include "hip/hip_runtime.h"
#include "cudaPart.cuh"

int readFile(const char *name, float **buf, int &numCh) {
	SF_INFO info;
	SNDFILE *sndfile;
	memset(&info, 0, sizeof(info));
	info.format = 0;
	sndfile = sf_open(name, SFM_READ, &info);
	if (sndfile == NULL) {
		fprintf(stderr, "ERROR. Cannot open %s\n", name);
		exit(1);
	}

	int size = info.frames;
	numCh = info.channels;

	*buf = (float*)malloc(sizeof(float) * size);

	if (info.channels == 1) {
		sf_read_float(sndfile, *buf, size);
	}

	else {
		/*Sum into mono & do RMS*/
		if (info.channels == 2) {
			/*Allocate temporary memory for wave file*/
			float *temp_buf = (float*)malloc(sizeof(float) * info.frames * 2);

			/*Read wave file into temporary memory*/
			sf_read_float(sndfile, temp_buf, info.frames * 2);

			/*Sum R & L*/
			for (int i = 0; i < info.frames; i++) {
				*buf[i] = temp_buf[i * 2] / 2.0 + temp_buf[i * 2 + 1] / 2.0;
			}

			free(temp_buf);

		}
		else {
			fprintf(stderr, "ERROR: %s : Only mono or stereo accepted", name);
		}
	}
	sf_close(sndfile);
	return size;
}

void cudaFFT(int argc, char **argv, Data *p) {

	std::string input = "Taiklatalvi.wav";
	std::string reverb = "s1_r1_b_441_mono.wav";
	if (argc == 2) {
		if (argv[1][1] != '>')
			input = argv[1];
	}
	if (argc == 3) {
		input = argv[1];
		reverb = argv[2];
	}

	float *ibuf, *rbuf;
	int SIGNAL_SIZE = 0, FILTER_KERNEL_SIZE = 0;

	fprintf(stderr, "Reading input file\n");
	int inputCh;
	SIGNAL_SIZE = readFile(input.c_str(), &ibuf, inputCh);

	fprintf(stderr, "Reading reverb file\n");
	FILTER_KERNEL_SIZE = readFile(reverb.c_str(), &rbuf, inputCh);
	if (inputCh != 1) {
		fprintf(stderr, "ERROR: Only mono reverb sources accepted");
		exit(2);
	}

	findCudaDevice(argc, (const char **)argv);

	fprintf(stderr, "Doing GPU Convolution\n");
	/*Pad signal and filter kernel to same length*/
	float *h_padded_signal;
	float *h_padded_filter_kernel;
	//new_size = SIGNAL_SIZE + (FILTER_KERNEL_SIZE) % 2
	int new_size = PadData(ibuf, &h_padded_signal, SIGNAL_SIZE,
		rbuf, &h_padded_filter_kernel, FILTER_KERNEL_SIZE);
	int mem_size = sizeof(float) * new_size;


	/*MOVING SIGNAL TO GPU*/
	// Allocate device memory for signal
	float *d_signal;
	checkCudaErrors(hipMalloc((void **)&d_signal, mem_size));

	// Copy signal from host to device
	checkCudaErrors(hipMemcpy(d_signal, h_padded_signal, mem_size,
		hipMemcpyHostToDevice));

	/*MOVING IMPULSE TO GPU*/
	// Allocate device memory for filter kernel
	float *d_filter_kernel;
	checkCudaErrors(hipMalloc((void **)&d_filter_kernel, mem_size));
	// Copy host memory to device
	checkCudaErrors(hipMemcpy(d_filter_kernel, h_padded_filter_kernel, mem_size,
		hipMemcpyHostToDevice));

	/*FIND RMS OF ORIGINAL SIGNAL*/
	/*Convert raw float pointer into a thrust device pointer*/
	thrust::device_ptr<float> thrust_d_signal(d_signal);

	/*Declare thrust operators*/
	square<float> unary_op;
	thrust::plus<float> binary_op;

	/*Perform thrust reduction to find rms*/
	float rms = std::sqrt(thrust::transform_reduce(thrust_d_signal, thrust_d_signal + new_size, unary_op, 0.0f, binary_op) / new_size);

	///////////////////////////////////////////////////////////////////////////////
	/*GPU PROCESSING*/
	///////////////////////////////////////////////////////////////////////////////

	// CUFFT plan simple API
	hipfftHandle plan;
	checkCudaErrors(hipfftPlan1d(&plan, new_size, HIPFFT_R2C, 1));
	hipfftHandle outplan;
	checkCudaErrors(hipfftPlan1d(&outplan, new_size, HIPFFT_C2R, 1));

	/*Create complex arrays*/
	hipfftComplex *d_sig_complex;
	checkCudaErrors(hipMalloc(&d_sig_complex, new_size * sizeof(hipfftComplex)));
	hipfftComplex *d_filter_complex;
	checkCudaErrors(hipMalloc(&d_filter_complex, new_size * sizeof(hipfftComplex)));

	/*FFT*/
	printf("Transforming signal hipfftExecR2C\n");
	checkCudaErrors(hipfftExecR2C(plan, (hipfftReal *)d_signal, d_sig_complex));
	checkCudaErrors(hipfftExecR2C(plan, (hipfftReal *)d_filter_kernel, d_filter_complex));

	/*CONVOLUTION*/
	// Multiply the coefficients together and normalize the result
	printf("Launching ComplexPointwiseMulAndScale<<< >>>\n");
	int blockSize = 256;
	int numBlocks = (new_size + blockSize - 1) / blockSize;
	ComplexPointwiseMulAndScale << < numBlocks, blockSize >> > (d_sig_complex, d_filter_complex, new_size, 1.0f / new_size);
	// Check if kernel execution generated and error
	getLastCudaError("Kernel execution failed [ ComplexPointwiseMulAndScale ]");

	/*IFFT*/
	// Transform signal back
	printf("Transforming signal back hipfftExecC2R\n");
	checkCudaErrors(hipfftExecC2R(outplan, d_sig_complex, d_signal));

	if (hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "Cuda error: failed to synchronize\n");
	}

	/*Find RMS of resulting signal*/
	thrust::device_ptr<float> thrust_d_output_signal2(d_signal);
	float rms2 = std::sqrt(thrust::transform_reduce(thrust_d_signal, thrust_d_signal + new_size, unary_op, 0.0f, binary_op) / new_size);
	printf("RMS1: %f RMS2: %f\n", rms, rms2);

	/*Scale resulting signal according to input signal*/
	MyFloatScale << < numBlocks, blockSize >> > (d_signal, new_size, rms / rms2);

	/*MOVE BACK TO CPU & STORE IN STRUCT*/
	float *obuf = (float*)malloc(sizeof(float) * new_size);
	checkCudaErrors(hipMemcpy(obuf, d_signal, new_size * sizeof(float), hipMemcpyDeviceToHost));
	p->buf = obuf;
	p->length = new_size;

	/*Store pointer to pointer of signal on device in struct*/
	//p->d_buf = &d_signal;

	fprintf(stderr, "Samples: %i\nTotal Bytes: %i\nTotal KB: %f3\nTotal MB: %f3\n\n\n", new_size, mem_size, mem_size / (float)1024, mem_size / (float)1024 / (float)1024);
	////////////////////////////////////////////////////////////////////////////////
	///*NOTE: GPU Convolution was not fast enough because of the large overhead
	//of FFT and IFFT. Keeping the code here for future purposes*/
	//
	/*Convolved signal on device*/
	//p->dbuf = d_signal;
	////////////////////////////////////////////////////////////////////////////////

	/*Write reverberated sound file*/
	//SndfileHandle file = SndfileHandle("output.wav", SFM_WRITE, isfinfo.format, isfinfo.channels, isfinfo.samplerate);
	//file.write(obuf, new_size);

	/*Destroy CUFFT context*/
	checkCudaErrors(hipfftDestroy(plan));
	checkCudaErrors(hipfftDestroy(outplan));

	/*Free memory*/

	free(ibuf);
	free(rbuf);

	free(h_padded_signal);
	free(h_padded_filter_kernel);

	checkCudaErrors(hipFree(d_signal));
	checkCudaErrors(hipFree(d_filter_kernel));
	checkCudaErrors(hipFree(d_sig_complex));
	checkCudaErrors(hipFree(d_filter_complex));


}