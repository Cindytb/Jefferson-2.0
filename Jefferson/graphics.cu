#include "hip/hip_runtime.h"
#include "graphics.cuh"

////////////////////////////////////////////////////////////////////////////////

// vbo variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;


double tan40 = tan(40.0 * PI / 180);
StopWatchInterface *timer = NULL;

float g_fAnim = 0.0;

//ball variables
float ball_x = 0.5, ball_y = 0.0, ball_z = 0.0;
float ball_rotate_x = 0.0, ball_rotate_y = 0.0, ball_rotate_z = 0.0;
float temp = 0.005f;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;

float translate_z = -3.0;
float translate_x = 0;

// Absolute rotation values (0-359 degrees) and rotation increments for each frame
double rotation_x = 0, rotation_x_increment = 0.1;
double rotation_y = 0, rotation_y_increment = 0.05;
double rotation_z = 0, rotation_z_increment = 0.03;

// Flag for rendering as lines or filled polygons
int filling = 1; //0=OFF 1=ON

				 //Lights settings
GLfloat light_ambient[] = { 0.3f, 0.3f, 0.3f, 0.3f };
GLfloat light_diffuse[] = { 0.2f, 0.2f, 0.2f, 0.2f };
GLfloat light_specular[] = { 0.2f, 0.2f, 0.2f, 0.2f };
GLfloat light_position[] = { 0.0f, 50.0f, 1.0f, 1.0f };

//Materials settings
GLfloat mat_ambient[] = { 0.1f, 0.1f, 0.1f, 0.0f };
GLfloat mat_diffuse[] = { -0.2f, -0.2f, -0.2f, -0.0f };
GLfloat mat_specular[] = { -0.2f, -0.2f, -0.2f, -0.0f };
GLfloat mat_shininess[] = { 0.01f };

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;
const char *sSDKsample = "Cindy Bui Final Project";
Data *GP;
std::stringstream s;


VBO *obj;

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////


void launch_kernel(float4 *pos, unsigned int mesh_width,
	unsigned int mesh_height, float time)
{
	// execute the kernel
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	simple_vbo_kernel << < grid, block >> > (pos, mesh_width, mesh_height, time);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int graphicsMain(int argc, char **argv, Data *p)
{
	char *ref_file = NULL;
	GP = p;
	pArgc = &argc;
	pArgv = argv;



#if defined(__linux__)
	setenv("DISPLAY", ":0", 0);
#endif

	printf("%s starting...\n", sSDKsample);

	if (argc > 1)
	{
		if (checkCmdLineFlag(argc, (const char **)argv, "file"))
		{
			// In this mode, we are running non-OpenGL and doing a compare of the VBO was generated correctly
			getCmdLineArgumentString(argc, (const char **)argv, "file", (char **)&ref_file);
		}
	}

	printf("\n");

	runTest(argc, argv, ref_file);

	printf("%s completed, returned %s\n", sSDKsample, (g_TotalErrors == 0) ? "OK" : "ERROR!");
	//exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
	return 0;
}

void computeFPS()
{
	frameCount++;
	fpsCount++;

	if (fpsCount == fpsLimit)
	{
		avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
		fpsCount = 0;
		fpsLimit = (int)MAX(avgFPS, 1.f);

		sdkResetTimer(&timer);
	}

	char fps[256];
	sprintf(fps, "Cuda GL Interop (VBO): %3.1f fps (Max 100Hz)", avgFPS);
	glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
	/*Old*/
	glutInit(argc, argv);
	//glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("Cindy Bui Final Project");
	glutDisplayFunc(display);
	glutIdleFunc(display);

	glutTimerFunc(REFRESH_DELAY, timerEvent, 0);

	// initialize necessary OpenGL extensions
	if (!isGLVersionSupported(2, 0))
	{
		fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		return false;
	}

	// default initialization
	//RGB: 193 215 229
	float red = 193.0f / 256.0f;
	float green = 215.0f / 256.0f;
	float blue = 229.0f / 256.0f;
	glClearColor(red, green, blue, 1.0);

	// viewport
	glViewport(0, 0, window_width, window_height);

	//Lights initialization and activation
	glLightfv(GL_LIGHT1, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT1, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT1, GL_SPECULAR, light_specular);
	glLightfv(GL_LIGHT1, GL_POSITION, light_position);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_LIGHT1);
	glEnable(GL_LIGHT2);
	glEnable(GL_LIGHT3);

	//Materials initialization and activation
	glMaterialfv(GL_FRONT, GL_AMBIENT, mat_ambient);
	glMaterialfv(GL_FRONT, GL_DIFFUSE, mat_diffuse);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);

	glEnable(GL_COLOR_MATERIAL);
	//Other initializations
	glShadeModel(GL_SMOOTH); // Type of shading for the polygons
	glHint(GL_PERSPECTIVE_CORRECTION_HINT, GL_NICEST); // Texture mapping perspective correction (OpenGL... thank you so much!)
	glEnable(GL_TEXTURE_2D); // Texture mapping ON
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL); // Polygon rasterization mode (polygon filled)
	glEnable(GL_CULL_FACE); // Enable the back face culling

	// projection
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(60.0, (GLfloat)window_width / (GLfloat)window_height, 0.1, 10.0);

	/*Attempting to create a face*/
	printf("...Loading body\n");
	ObjLoad("body.3ds");
	printf("...Loading eyes\n");
	ObjLoad("eyes.3ds");
	printf("...Loading smile\n");
	ObjLoad("smile.3ds");
	printf("...Loading letter\n");
	ObjLoad("letter.3ds");
	printf("...Loading hat\n");
	ObjLoad("hat.3ds");
	glewInit();
	SDK_CHECK_ERROR_GL();

	return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char **argv, char *ref_file)
{

	// Create the CUTIL timer
	sdkCreateTimer(&timer);
	// First initialize OpenGL context, so we can properly set the GL for CUDA.
	// This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
	if (false == initGL(&argc, argv))
	{
		return false;
	}
	// register callbacks
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMouseFunc(mouse);
	glutMotionFunc(motion);
	glutSpecialFunc(specialKeys);
#if defined (__APPLE__) || defined(MACOSX)
	atexit(cleanup);
#else
	glutCloseFunc(cleanup);
#endif
	
#if(DEBUGMODE != 1)
	/*MOVING SIGNAL TO GPU*/
	// Allocate device memory for signal
	float *d_signal;
	checkCudaErrors(hipMalloc((void **)&d_signal, GP->length * sizeof(float)));

	// Copy signal from host to device
	checkCudaErrors(hipMemcpy(d_signal, GP->buf, GP->length * sizeof(float),
		hipMemcpyHostToDevice));

	obj = new VBO(&d_signal, &translate_x, GP->length, 1 / 44100.0f);
	obj->init();
	obj->averageNum = 100;
	obj->create();
#endif
	// create sine wave VBO
	createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

	// run the cuda part
	runCuda(&cuda_vbo_resource);

	// start rendering mainloop
	glutMainLoop();

	return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource)
{
	// map OpenGL buffer object for writing from CUDA
	float4 *dptr;
	checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
	size_t num_bytes;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
		*vbo_resource));

	launch_kernel(dptr, mesh_width, mesh_height, g_fAnim);

	// unmap buffer object
	checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

#ifdef _WIN32
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) fopen_s(&fHandle, filename, mode)
#endif
#else
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) (fHandle = fopen(filename, mode))
#endif
#endif


////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
	unsigned int vbo_res_flags)
{
	assert(vbo);

	// create buffer object
	glGenBuffers(1, vbo);
	glBindBuffer(GL_ARRAY_BUFFER, *vbo);

	// initialize buffer object
	unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

	glBindBuffer(GL_ARRAY_BUFFER, 0);

	// register this buffer object with CUDA
	checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

	//SDK_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

	// unregister this buffer object with CUDA
	checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

	glBindBuffer(1, *vbo);
	glDeleteBuffers(1, vbo);

	*vbo = 0;
}
void moveBar(Data p) {
	if (p.pauseStatus == true) {
		return;
	}
	translate_x = (float)p.count * -(obj->ratio);
}
////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
	sdkStartTimer(&timer);

	// run CUDA kernel to generate vertex positions
	runCuda(&cuda_vbo_resource);

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// set view matrix
	glMatrixMode(GL_MODELVIEW);

	glLoadIdentity();
	/*zooming in & out*/
	glTranslatef(0.0, 0.0, translate_z);

	/*Rotating around the mesh's axis*/
	glRotatef(rotate_x, 1.0, 0.0, 0.0);
	glRotatef(rotate_y, 0.0, 1.0, 0.0);
	moveBar(*GP);
#if(DEBUGMODE != 1)
	/*Calculate the radius, distance, elevation, and azimuth*/
	float r = std::sqrt(ball_x * ball_x + ball_z * ball_z + ball_y * ball_y);
	float horizR = std::sqrt(ball_x * ball_x + ball_z * ball_z);
	float ele = (float)atan2(ball_y, horizR) * 180.0f / PI;
	//s.str(std::string());
	float obj_azi = (float)atan2(ball_x / r, ball_z / r) * 180.0f / PI;
	/*s << "Azimuth: " << obj_azi;
	s << "Elevation: " << ele;
	s << "Radius: " << r;*/
	GP->hrtf_idx = pick_hrtf(ele, obj_azi);
	float newR = r / 100 + 1;
	GP->gain = 1 / pow(newR, 2);
	
	float rotateVBO_y = (float)atan2(-ball_z, ball_x) * 180.0f / PI;

	if (rotateVBO_y < 0) {
		rotateVBO_y += 360;
	}
	//printf("x: %3f\ty: %3f\tz: %3f\tX: %3f\tY: %3f\tZ: %3f\n", ball_x, ball_y, ball_z, rotateVBO_x, rotateVBO_y, rotateVBO_z);
#endif
	obj->averageNum = 100;
	obj->update();
	obj->draw(rotateVBO_y, ele, 0.0f);

	// render from the vbo
	glBindBuffer(GL_ARRAY_BUFFER, vbo);
	glVertexPointer(4, GL_FLOAT, 0, 0);
	glEnableClientState(GL_VERTEX_ARRAY);

	/*SINE WAVE COLORS*/
	glColor3f( 47.0f/256.0f, 63.0f/256.0f, 45.0f/256.0f );
	glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
	glDisableClientState(GL_VERTEX_ARRAY);

	
	// Enable depth test
	glEnable(GL_DEPTH_TEST);
	GLUquadricObj *quadric;
	quadric = gluNewQuadric();
	gluQuadricTexture(quadric, GL_TRUE);
	gluQuadricNormals(quadric, GLU_SMOOTH);

	/*IT'S JEFFERSON*/
	for (int i = 0; i<obj_qty; i++)
	{
		//glTranslatef(0.0, 0.0, 0.0);
		if(i == 0) glColor3f(60.0f / 256.0f, 52.0f / 256.0f, 96.0f / 256.0f);
		else if (i == 1) glColor3f(0.0f, 1.0f, 1.0f);
		else if (i == 4) glColor3f(0.0f, 0.0f, 0.0f);
		else glColor3f(1.0f, 1.0f, 1.0f);
		glPushMatrix(); // We save the current matrix
		glScalef(0.5f, 0.5f, 0.5f);
		//glTranslatef(ball_x, ball_y, ball_z);
		glTranslatef(0.0, 0.0, 0.0);
		glMultMatrixf(&object[i].matrix[0][0]); // Now let's multiply the object matrix by the identity-first matrix

		if (object[i].id_texture != -1)
		{
			glBindTexture(GL_TEXTURE_2D, object[i].id_texture); // We set the active texture 
			glEnable(GL_TEXTURE_2D); // Texture mapping ON
		}
		else
			glDisable(GL_TEXTURE_2D); // Texture mapping OFF

		glBegin(GL_TRIANGLES); // glBegin and glEnd delimit the vertices that define a primitive (in our case triangles)
		for (int j = 0; j<object[i].polygons_qty; j++)
		{
			//----------------- FIRST VERTEX -----------------
			//Normal coordinates of the first vertex
			glNormal3f(object[i].normal[object[i].polygon[j].a].x,
				object[i].normal[object[i].polygon[j].a].y,
				object[i].normal[object[i].polygon[j].a].z);
			// Texture coordinates of the first vertex
			glTexCoord2f(object[i].mapcoord[object[i].polygon[j].a].u,
				object[i].mapcoord[object[i].polygon[j].a].v);
			// Coordinates of the first vertex
			glVertex3f(object[i].vertex[object[i].polygon[j].a].x,
				object[i].vertex[object[i].polygon[j].a].y,
				object[i].vertex[object[i].polygon[j].a].z);

			//----------------- SECOND VERTEX -----------------
			//Normal coordinates of the second vertex
			glNormal3f(object[i].normal[object[i].polygon[j].b].x,
				object[i].normal[object[i].polygon[j].b].y,
				object[i].normal[object[i].polygon[j].b].z);
			// Texture coordinates of the second vertex
			glTexCoord2f(object[i].mapcoord[object[i].polygon[j].b].u,
				object[i].mapcoord[object[i].polygon[j].b].v);
			// Coordinates of the second vertex
			glVertex3f(object[i].vertex[object[i].polygon[j].b].x,
				object[i].vertex[object[i].polygon[j].b].y,
				object[i].vertex[object[i].polygon[j].b].z);

			//----------------- THIRD VERTEX -----------------
			//Normal coordinates of the third vertex
			glNormal3f(object[i].normal[object[i].polygon[j].c].x,
				object[i].normal[object[i].polygon[j].c].y,
				object[i].normal[object[i].polygon[j].c].z);
			// Texture coordinates of the third vertex
			glTexCoord2f(object[i].mapcoord[object[i].polygon[j].c].u,
				object[i].mapcoord[object[i].polygon[j].c].v);
			// Coordinates of the Third vertex
			glVertex3f(object[i].vertex[object[i].polygon[j].c].x,
				object[i].vertex[object[i].polygon[j].c].y,
				object[i].vertex[object[i].polygon[j].c].z);

		}
		glEnd();
		glPopMatrix(); // Restore the previous matrix 
	}

	/*SOUND SOURCE SPHERE*/
	//Source RGB: 119, 207, 131
	float red = 119.0f / 256.0f;
	float green = 207.0f / 256.0f;
	float blue = 131.0f / 256.0f;
	glColor3f(red, green, blue);
	glPushMatrix();
	glTranslatef(ball_x, ball_y, ball_z);
	gluSphere(quadric, 0.1, 20, 50);
	glPopMatrix();


	/*GL Setup to display text onto the screen for debugging purposes*/
	//glMatrixMode(GL_PROJECTION);
	//glPushMatrix();
	//glLoadIdentity();
	//glMatrixMode(GL_MODELVIEW);
	//glPushMatrix();
	//glLoadIdentity();
	//glDisable(GL_DEPTH_TEST);

	//glColor3f(255, 255, 255);
	//glRasterPos2f(0,0);
	//std::string temp = s.str();
	//int len = (int) temp.length();
	//for (int i = 0; i < len; i++) {
	//	glutBitmapCharacter(GLUT_BITMAP_TIMES_ROMAN_24, temp.at(i));
	//}
	//glEnable(GL_DEPTH_TEST); // Turn depth testing back on

	//glMatrixMode(GL_PROJECTION);
	//glPopMatrix(); // revert back to the matrix I had before.
	//glMatrixMode(GL_MODELVIEW);
	//glPopMatrix();

	/*Step up the phase for the sinusoidal waves*/
	g_fAnim += 0.01f;

	/*Push out the OpenGL buffer*/
	glutSwapBuffers();
	sdkStopTimer(&timer);
	computeFPS();
}
void timerEvent(int value){
	if (glutGetWindow())
	{
		glutPostRedisplay();
		glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
	}
}
void cleanup()
{
	sdkDeleteTimer(&timer);

	if (vbo)
	{
		deleteVBO(&vbo, cuda_vbo_resource);
	}
#if(DEBUGMODE != 1)
	/*Close output file*/
	sf_close(GP->sndfile);

	/* Stop stream */
	closePA();
	

#endif
}
////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	float dist = std::sqrt(ball_x * ball_x + ball_z * ball_z);

	switch (key)
	{
	case('r'):
		rotate_x = 0.0f;
		rotate_y = 0.0f;
		translate_z = -3.0f;
		ball_x = 0.5;
		ball_y = 0.0;
		ball_z = 0.0;
		break;
	case('w'):
		//value is 40 degrees in radians
		if (ball_y >= 0 || ball_y < 0 && (atan(ball_y / dist) < 0.6981317))
			ball_y += temp;
		break;
	case('s'):
		ball_y -= temp;
		break;
		/*TODO: Fix this logic*/
	case('a'):
		if (ball_y >= 0 || ball_y < 0 && (atan(ball_y / std::sqrt(pow(ball_x - temp, 2) + pow(ball_z, 2)) < tan40)))
			ball_x -= temp;
		break;
	case('d'):
		if (ball_y >= 0 || ball_y < 0 && (atan(ball_y / std::sqrt(pow(ball_x + temp, 2) + pow(ball_z, 2)) < tan40)))
			ball_x += temp;
		break;
	case (27):
		printf("Finished playout\n");
#if defined(__APPLE__) || defined(MACOSX)
		exit(EXIT_SUCCESS);
#else
		glutDestroyWindow(glutGetWindow());
		return;
#endif
	}
}
void specialKeys(int key, int x, int y) {

	switch (key) {
	case GLUT_KEY_LEFT:
		if (ball_y <= 0 || ball_y > 0 && (atan(ball_y / std::sqrt(pow(ball_x - temp, 2) + pow(ball_z, 2)) < tan40)))
			ball_x -= temp;
		break;
	case GLUT_KEY_RIGHT:
		if (ball_y <= 0 || ball_y > 0 && (atan(ball_y / std::sqrt(pow(ball_x + temp, 2) + pow(ball_z, 2)) < tan40)))
			ball_x += temp;
		break;
	case GLUT_KEY_UP:
		if (ball_y <= 0 || ball_y > 0 && (atan(ball_y / std::sqrt(pow(ball_x, 2) + pow(ball_z - temp, 2)) < tan40)))
			ball_z -= temp;
		break;
	case GLUT_KEY_DOWN:
		if (ball_y <= 0 || ball_y > 0 && (atan(ball_y / std::sqrt(pow(ball_x, 2) + pow(ball_z + temp, 2)) < tan40)))
			ball_z += temp;
		break;
	}
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
	if (state == GLUT_DOWN)
	{
		mouse_buttons |= 1 << button;
		if (button == 3)
		{
			translate_z += 0.1f;
		}
		else if (button == 4)
		{
			translate_z -= 0.1f;
		}
		//printf("Scroll %s At %d %d\n", (button == 3) ? "Up" : "Down", x, y);
	}
	else if (state == GLUT_UP)
	{
		mouse_buttons = 0;
	}
	mouse_old_x = x;
	mouse_old_y = y;
}
/*Rotate the perspective*/
void motion(int x, int y)
{
	float dx, dy;
	dx = (float)(x - mouse_old_x);
	dy = (float)(y - mouse_old_y);

	if (mouse_buttons & 1)
	{
		rotate_x += dy * 0.2f;
		rotate_y += dx * 0.2f;
	}
	else if (mouse_buttons & 4)
	{
		translate_z += dy * 0.01f;
	}
	mouse_old_x = x;
	mouse_old_y = y;
}