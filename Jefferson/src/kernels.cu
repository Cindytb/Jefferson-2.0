#include "hip/hip_runtime.h"
#include "kernels.cuh"

struct is_negative {
	__host__ __device__ bool operator()(float x) {
		if (x < 0) {
			return true;
		}
		return false;
	}
};
template <typename T>
struct square
{
	__host__ __device__
		T operator()(const T& x) const
	{
		return x * x;
	}
};
///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float4 *pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	// calculate uv coordinates
	float u = x / (float)width;
	float v = y / (float)height;
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;

	// calculate simple sine wave pattern
	float freq = 4.0f;
	float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

	// write output vertex
	pos[y*width + x] = make_float4(u, -0.55f, v, 1.0f);
}

// hipfftComplex pointwise multiplication
__global__ void ComplexPointwiseMulAndScale(hipfftComplex *a, const hipfftComplex *b, int size, float scale)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < size; i += numThreads)
	{
		a[i] = cufftComplexScale(cufftComplexMul(a[i], b[i]), scale);
	}
}
__global__ void ComplexPointwiseMulAndScaleOutPlace(const hipfftComplex* a, const hipfftComplex* b, hipfftComplex* c, int size, float scale)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < size; i += numThreads)
	{
		c[i] = cufftComplexScale(cufftComplexMul(a[i], b[i]), scale);
	}
}
/*
	R(r) = (1 / (1 + (fs / vs) (r - r0)^2) ) * e^ ((-j2PI (fs/vs) * (r - r0) *k) / N)
			|----------FRAC-----------------|	  |------------exponent--------------|

	FRAC * e^(exponent)
	FRAC * (cosine(exponent) - sine(exponent))
	R[r].x = cosine(exponent) / FRAC
	R[r].y = -sine(exponent) / FRAC
	*/
__global__ void generateDistanceFactor(hipfftComplex *in, float frac, float fsvs, float r, int N){
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < N; i += numThreads)
	{
		in[i].x = cos(2 * PI * fsvs * r * i / N) / frac;
		in[i].y = -sin(2 * PI * fsvs * r * i / N) / frac;
	}
}

/*
	f[n] = n / (N - 1)
	g[n] = 1 - f[n]
*/
__global__ void crossFade(float* out1, float* out2, int numFrames){
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = threadID; i < numFrames; i += numThreads)
	{
		out1[i * 2] = out1[i * 2] * float(i) / (numFrames - 1) + out2[i * 2] * (1 - float(i) / (numFrames - 1));
		out1[i * 2 + 1] = out1[i * 2 + 1] * float(i) / (numFrames - 1) + out2[i * 2 + 1] * (1 - float(i) / (numFrames - 1));
	}

}
// hipfftComplex pointwise multiplication
__global__ void ComplexPointwiseMulInPlace(const hipfftComplex* in, hipfftComplex* out, int size) {
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < size; i += numThreads)
	{
		out[i] = cufftComplexMul(out[i], in[i]);
	}
}
__global__ void ComplexPointwiseAdd(hipfftComplex* in, hipfftComplex* out, int size)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < size; i += numThreads)
	{

		atomicAdd(&(out[i].x), in[i].x);
		atomicAdd(&(out[i].y), in[i].y);
		// out[i].x += in[i].x;
		// out[i].y += in[i].y;
	}
}

__global__ void timeDomainConvolutionNaive(float* ibuf, float* rbuf, float* obuf, long long oframes,
	long long rframes, int ch, float gain) {
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	float value = 0;
	for (int k = 0; k < rframes; k++) {
		value += ibuf[threadID - k] * rbuf[k];
	}
	obuf[threadID * 2 + ch] = value * gain;

}
__global__ void interleave(float* input, float* output, int size) {
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < size; i += numThreads) {
		output[2 * i] = input[i];
		output[2 * i + 1] = input[size + 2 + i];
	}
}

// hipfftComplex pointwise multiplication
__global__ void ComplexPointwiseMul(hipfftComplex* a, const hipfftComplex* b, hipfftComplex* c, int size){
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < size; i += numThreads)
	{
		c[i] = cufftComplexMul(a[i], b[i]);
	}
}
__global__ void MyFloatScale(float *a, int size, float scale) {
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < size; i += numThreads)
	{
		a[i] = a[i] * scale;
	}
}


// Pad data
int PadData(const float *signal, float **padded_signal, int signal_size,
	const float *filter_kernel, float **padded_filter_kernel, int filter_kernel_size)
{
	int minRadius = filter_kernel_size / 2;
	int maxRadius = filter_kernel_size - minRadius;
	int new_size = signal_size + maxRadius;

	// Pad signal
	float *new_data = (float *)malloc(sizeof(float) * new_size);
	memcpy(new_data + 0, signal, signal_size * sizeof(float));
	memset(new_data + signal_size, 0, (new_size - signal_size) * sizeof(float));
	*padded_signal = new_data;

	// Pad filter
	new_data = (float *)malloc(sizeof(float) * new_size);
	memcpy(new_data + 0, filter_kernel, filter_kernel_size * sizeof(float));
	memset(new_data + filter_kernel_size, 0, (new_size - filter_kernel_size) * sizeof(float));
	*padded_filter_kernel = new_data;
	return new_size;
}

// hipfftComplex scale
__device__ __host__ inline hipfftComplex cufftComplexScale(hipfftComplex a, float s)
{
	hipfftComplex c;
	c.x = s * a.x;
	c.y = s * a.y;
	return c;
}
// hipfftComplex multiplication
__device__ __host__ inline hipfftComplex cufftComplexMul(hipfftComplex a, hipfftComplex b)
{
	hipfftComplex c;
	c.x = a.x * b.x - a.y * b.y;
	c.y = a.x * b.y + a.y * b.x;
	return c;
}


__global__ void averagingKernel(float4 *pos, float *d_buf, unsigned int size, double ratio, int averageSize) {

	unsigned long modNum = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned long samp_num = modNum * averageSize;
	if (samp_num < size) {

		int end;
		if (size < samp_num + averageSize - 1) {
			end = size;
		}
		else {
			end = samp_num + averageSize - 1;
		}
		thrust::negate<float> op;
		thrust::transform_if(thrust::device, d_buf + samp_num, d_buf + end, d_buf + samp_num, op, is_negative());
		float avg = thrust::reduce(thrust::device, d_buf + samp_num, d_buf + end, 0.0f, thrust::plus<float>());
		avg /= (float)averageSize;

		float x = (float)samp_num * ratio;
		/*Flat 2D waveform for testing*/
		pos[modNum * 2] = make_float4(x, avg, 0, 1.0f);
		pos[modNum * 2 + 1] = make_float4(x, -avg, 0, 1.0f);

	}
}

__global__ void fill_kernel(thrust::device_ptr<float> dev_ptr, long long old_size, long long new_size)
{
	thrust::fill(dev_ptr + old_size, dev_ptr + new_size, (float)0.0f);
}

__global__ void fillZeros(float* buf, int size) {
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < size; i += numThreads)
	{
		buf[i] = 0.0f;
	}
}
void fillWithZeroes(float** target_buf, long long old_size, long long new_size, hipStream_t s) {
	thrust::device_ptr<float> dev_ptr(*target_buf);
	fill_kernel << <1, 1, 0, s >> > (dev_ptr, old_size, new_size);
}
void fillWithZeroes(float** target_buf, long long old_size, long long new_size) {
	thrust::device_ptr<float> dev_ptr(*target_buf);
	fill_kernel << <1, 1>> > (dev_ptr, old_size, new_size);
}
void fillWithZeroesKernel(float* buf, int size, hipStream_t s) {
	int numThreads = 256;
	int numBlocks = (size + numThreads - 1) / numThreads;
	fillZeros << < numThreads, numBlocks, 0, s >> > (buf, size);
}