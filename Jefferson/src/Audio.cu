#include "Audio.cuh"
#include <hip/hip_runtime.h>
PaStream *stream;

void initializePA(int fs) {
	PaError err;
	fprintf(stderr, "\n\n\n");
#if DEBUG != 1
	/*PortAudio setup*/
	PaStreamParameters outputParams;
	PaStreamParameters inputParams;

	/* Initializing PortAudio */
	err = Pa_Initialize();
	if (err != paNoError) {
		printf("PortAudio error: %s\n", Pa_GetErrorText(err));
		printf("\nExiting.\n");
		fprintf(stderr, "PortAudio error: %s\n", Pa_GetErrorText(err));
		fprintf(stderr, "\nExiting.\n");
		exit(1);
	}

	/* Input stream parameters */
	inputParams.device = Pa_GetDefaultInputDevice();
	inputParams.channelCount = 1;
	inputParams.sampleFormat = paFloat32;
	inputParams.suggestedLatency =
		Pa_GetDeviceInfo(inputParams.device)->defaultLowInputLatency;
	inputParams.hostApiSpecificStreamInfo = NULL;

	/* Ouput stream parameters */
	outputParams.device = Pa_GetDefaultOutputDevice();
	outputParams.channelCount = 2;
	outputParams.sampleFormat = paFloat32;
	outputParams.suggestedLatency =
		Pa_GetDeviceInfo(outputParams.device)->defaultLowOutputLatency;
	outputParams.hostApiSpecificStreamInfo = NULL;

	/* Open audio stream */
	err = Pa_OpenStream(&stream,
		&inputParams, /* no input */
		&outputParams,
		fs, FRAMES_PER_BUFFER,
		paNoFlag, /* flags */
		paCallback,
		&data);

	if (err != paNoError) {
		printf("PortAudio error: open stream: %s\n", Pa_GetErrorText(err));
		printf("\nExiting.\n");
		fprintf(stderr, "PortAudio error: open stream: %s\n", Pa_GetErrorText(err));
		fprintf(stderr, "\nExiting.\n");
		exit(1);
	}

	/* Start audio stream */
	err = Pa_StartStream(stream);
	if (err != paNoError) {
		printf("PortAudio error: start stream: %s\n", Pa_GetErrorText(err));
		printf("\nExiting.\n");
		fprintf(stderr, "PortAudio error: start stream: %s\n", Pa_GetErrorText(err));
		fprintf(stderr, "\nExiting.\n");
		exit(1);
	}
#endif

}

void closePA() {
	PaError err;
#if DEBUG != 1
	/* Stop stream */
	err = Pa_StopStream(stream);
	if (err != paNoError) {
		printf("PortAudio error: stop stream: %s\n", Pa_GetErrorText(err));
		printf("\nExiting.\n");
		fprintf(stderr, "PortAudio error: stop stream: %s\n", Pa_GetErrorText(err));
		fprintf(stderr, "\nExiting.\n");
		exit(1);
	}

	/* Close stream */
	err = Pa_CloseStream(stream);
	if (err != paNoError) {
		printf("PortAudio error: close stream: %s\n", Pa_GetErrorText(err));
		printf("\nExiting.\n");
		fprintf(stderr, "PortAudio error: close stream: %s\n", Pa_GetErrorText(err));
		fprintf(stderr, "\nExiting.\n");
		exit(1);
	}

	/* Terminate PortAudio */
	err = Pa_Terminate();
	if (err != paNoError) {
		printf("PortAudio error: terminate: %s\n", Pa_GetErrorText(err));
		printf("\nExiting.\n");
		fprintf(stderr, "PortAudio error: terminate: %s\n", Pa_GetErrorText(err));
		fprintf(stderr, "\nExiting.\n");
		exit(1);
	}
#endif
}
void callback_func(float *output, Data *p){
	// printf("%i\n", p->count);
	// for(int i = 0; i < 8; i++){
	// 	fprintf(stderr, "Stream No:%i - %s\n", i, hipStreamQuery(p->streams[i]) ? "Not Finished" : "Finished");
	// }
	// fprintf(stderr, "\n");
	checkCudaErrors(hipStreamSynchronize(p->streams[(p->blockNo - 2) % 5 * 2]));
	/*Copy into p->x pinned memory*/
	if (p->count + FRAMES_PER_BUFFER < p->length){
		memcpy(p->x + HRTF_LEN - 1, p->buf + p->count, FRAMES_PER_BUFFER * sizeof(float));
		p->count += FRAMES_PER_BUFFER;
	}
	else{
		int rem = p->length - p->count;
		memcpy(p->x + HRTF_LEN - 1, p->buf + p->count, rem * sizeof(float));
		memcpy(p->x + HRTF_LEN - 1 + rem, p->buf, (FRAMES_PER_BUFFER - rem) * sizeof(float));
		p->count = FRAMES_PER_BUFFER - rem;
	}

	// if (p->blockNo == 5) {
	// 	printf("%i\n", hipCtxPushCurrent(0));
	// }
	// fprintf(stderr, "Stream %i %s\n", p->blockNo % 5, hipStreamQuery(p->streams[p->blockNo % 5 * 2]) ? "Unfinished":"Finished");
	/*Enable pausing of audio*/
	if (p->pauseStatus == true) {
		for (int i = 0; i < FRAMES_PER_BUFFER; i++) {
			output[2 * i] = 0;
			output[2 * i + 1] = 0;
		}
		return;
	}
	memcpy(output, p->intermediate, FRAMES_PER_BUFFER * 2 * sizeof(float));
	// fprintf(stderr, "%i %i %i %i %i\n", p->blockNo % 5, (p->blockNo - 1) % 5, (p->blockNo - 2) % 5, (p->blockNo - 3) % 5, (p->blockNo - 4) % 5);
	/*Send*/
	checkCudaErrors(hipMemcpyAsync(p->d_input[p->blockNo % 5], p->x, COPY_AMT * sizeof(float), hipMemcpyHostToDevice, p->streams[(p->blockNo) % 5 * 2]));
	/*Process*/
	GPUconvolve_hrtf(p->d_input[(p->blockNo - 1) % 5] + HRTF_LEN, p->hrtf_idx, p->d_output[(p->blockNo - 1) % 5], FRAMES_PER_BUFFER, p->gain, &(p->streams[(p->blockNo - 1) % 5 * 2]));
	/*Idle blockNo - 2*/
	/*Idle blockNo - 3*/
	/*Return & fill intermediate*/
	checkCudaErrors(hipMemcpyAsync(p->intermediate, p->d_output[(p->blockNo - 4) % 5], FRAMES_PER_BUFFER * 2 * sizeof(float), hipMemcpyDeviceToHost, p->streams[(p->blockNo - 3) % 5 * 2]));

	
	/*Overlap-save*/
	memcpy(p->x, p->x + FRAMES_PER_BUFFER, (HRTF_LEN - 1) * sizeof(float));
	p->blockNo++;

	//sf_writef_float(p->sndfile, output, framesPerBuffer);
	return;
}
static int paCallback(const void *inputBuffer, void *outputBuffer,
	unsigned long framesPerBuffer,
	const PaStreamCallbackTimeInfo* timeInfo,
	PaStreamCallbackFlags statusFlags,
	void *userData)
{
	/* Cast data passed through stream to our structure. */
	Data *p = (Data *)userData;
	float *output = (float *)outputBuffer;
	callback_func(output, p);
	return 0;
}

