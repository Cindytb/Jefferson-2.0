#include "hip/hip_runtime.h"
#include "Audio.cuh"

PaStream *stream;

void initializePA(int fs) {
	PaError err;
	fprintf(stderr, "\n\n\n");
#if DEBUG != 1
	/*PortAudio setup*/
	PaStreamParameters outputParams;
	PaStreamParameters inputParams;

	/* Initializing PortAudio */
	err = Pa_Initialize();
	if (err != paNoError) {
		printf("PortAudio error: %s\n", Pa_GetErrorText(err));
		printf("\nExiting.\n");
		fprintf(stderr, "PortAudio error: %s\n", Pa_GetErrorText(err));
		fprintf(stderr, "\nExiting.\n");
		exit(1);
	}

	/* Input stream parameters */
	inputParams.device = Pa_GetDefaultInputDevice();
	inputParams.channelCount = 1;
	inputParams.sampleFormat = paFloat32;
	inputParams.suggestedLatency =
		Pa_GetDeviceInfo(inputParams.device)->defaultLowInputLatency;
	inputParams.hostApiSpecificStreamInfo = NULL;

	/* Ouput stream parameters */
	outputParams.device = Pa_GetDefaultOutputDevice();
	outputParams.channelCount = 2;
	outputParams.sampleFormat = paFloat32;
	outputParams.suggestedLatency =
		Pa_GetDeviceInfo(outputParams.device)->defaultLowOutputLatency;
	outputParams.hostApiSpecificStreamInfo = NULL;

	/* Open audio stream */
	err = Pa_OpenStream(&stream,
		&inputParams, /* no input */
		&outputParams,
		fs, FRAMES_PER_BUFFER,
		paNoFlag, /* flags */
		paCallback,
		&data);

	if (err != paNoError) {
		printf("PortAudio error: open stream: %s\n", Pa_GetErrorText(err));
		printf("\nExiting.\n");
		fprintf(stderr, "PortAudio error: open stream: %s\n", Pa_GetErrorText(err));
		fprintf(stderr, "\nExiting.\n");
		exit(1);
	}

	/* Start audio stream */
	err = Pa_StartStream(stream);
	if (err != paNoError) {
		printf("PortAudio error: start stream: %s\n", Pa_GetErrorText(err));
		printf("\nExiting.\n");
		fprintf(stderr, "PortAudio error: start stream: %s\n", Pa_GetErrorText(err));
		fprintf(stderr, "\nExiting.\n");
		exit(1);
	}
#endif

}

void closePA() {
	PaError err;
#if DEBUG != 1
	/* Stop stream */
	err = Pa_StopStream(stream);
	if (err != paNoError) {
		printf("PortAudio error: stop stream: %s\n", Pa_GetErrorText(err));
		printf("\nExiting.\n");
		fprintf(stderr, "PortAudio error: stop stream: %s\n", Pa_GetErrorText(err));
		fprintf(stderr, "\nExiting.\n");
		exit(1);
	}

	/* Close stream */
	err = Pa_CloseStream(stream);
	if (err != paNoError) {
		printf("PortAudio error: close stream: %s\n", Pa_GetErrorText(err));
		printf("\nExiting.\n");
		fprintf(stderr, "PortAudio error: close stream: %s\n", Pa_GetErrorText(err));
		fprintf(stderr, "\nExiting.\n");
		exit(1);
	}

	/* Terminate PortAudio */
	err = Pa_Terminate();
	if (err != paNoError) {
		printf("PortAudio error: terminate: %s\n", Pa_GetErrorText(err));
		printf("\nExiting.\n");
		fprintf(stderr, "PortAudio error: terminate: %s\n", Pa_GetErrorText(err));
		fprintf(stderr, "\nExiting.\n");
		exit(1);
	}
#endif
}

static int paCallback(const void *inputBuffer, void *outputBuffer,
	unsigned long framesPerBuffer,
	const PaStreamCallbackTimeInfo* timeInfo,
	PaStreamCallbackFlags statusFlags,
	void *userData)
{
	/* Cast data passed through stream to our structure. */
	Data *p = (Data *)userData;
	float *output = (float *)outputBuffer;
	//float *input = (float *)inputBuffer; /* input not used in this code */
	float *px;
	unsigned int i;
	float *buf = (float*)malloc(sizeof(float) * 2 * framesPerBuffer - HRTF_LEN);

	/*CPU/RAM Copy data loop*/
	for (int i = 0; i < framesPerBuffer; i++) {
		p->x[HRTF_LEN - 1 + i] = p->buf[p->count];
		p->count++;
		if (p->count == p->length) {
			p->count = 0;
		}
	}
	/*convolve with HRTF on CPU*/
	convolve_hrtf(&p->x[HRTF_LEN], p->hrtf_idx, output, framesPerBuffer, p->gain);

	/*Enable pausing of audio*/
	if (p->pauseStatus == true) {
		for (i = 0; i < framesPerBuffer; i++) {
			output[2 * i] = 0;
			output[2 * i + 1] = 0;
		}
		return 0;
	}

	////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////
	/*NOTE: GPU Convolution was not fast enough because of the large overhead
	of FFT and IFFT. Keeping the code here for future purposes*/
	/*CUDA Copy*/
	//hipDeviceSynchronize();
	//int blockSize = 256;
	//int numBlocks = (framesPerBuffer + blockSize - 1) / blockSize;
	//if(p->count + framesPerBuffer <= p->length) {
	//	copyMe << < numBlocks, blockSize >> > (framesPerBuffer, p->d_x, &p->dbuf[p->count]);
	//	hipDeviceSynchronize();
	//	p->count += framesPerBuffer;
	//}
	//
	//else {
	//	int remainder = p->length - p->count - framesPerBuffer;
	//	copyMe << < numBlocks, blockSize >> > (p->length - p->count, p->d_x, &p->dbuf[p->count]);
	//	p->count = 0;
	//	copyMe << < numBlocks, blockSize >> > (remainder, p->d_x, &p->dbuf[p->count]);
	//	p->count += remainder;
	//}
	/*Convolve on GPU*/
	//GPUconvolve_hrtf(p->d_x, framesPerBuffer, p->hrtf_idx, output, framesPerBuffer, p->gain);
	////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////


	/* copy last HRTF_LEN-1 samples of x data to "history" part of x for use next time */
	px = p->x;
	for (i = 0; i<HRTF_LEN - 1; i++) {
		px[i] = px[framesPerBuffer + i];
	}
	//sf_writef_float(p->sndfile, output, framesPerBuffer);
	return 0;
}

