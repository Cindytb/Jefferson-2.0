#include "hip/hip_runtime.h"
#include "cudaPart.cuh"
struct is_negative {
	__host__ __device__ bool operator()(float x) {
		if (x < 0) {
			return true;
		}
		return false;
	}
};
template <typename T>
struct square
{
	__host__ __device__
		T operator()(const T& x) const
	{
		return x * x;
	}
};
typedef float2 Complex; 
const bool reverbFlag = false;
int readFile(const char *name, float **buf, int &numCh) {
	SndfileHandle file = SndfileHandle(name);
	int size = file.frames();
	numCh = file.channels();

	*buf = (float*)malloc(sizeof(float) * size);

	if (numCh == 1) {
		size_t count = file.readf(*buf, size);
		if (count != size) {
			fprintf(stderr, "ERROR. Cannot read all of %s\n", name);
			exit(1);
		}
	}

	else {
		/*Sum into mono & do RMS*/
		if (numCh == 2) {
			/*Allocate temporary memory for wave file*/
			float *temp_buf = new float[size * 2];

			/*Read wave file into temporary memory*/
			size_t count = file.readf(temp_buf, size);
			if (count != size) {
				fprintf(stderr, "ERROR. Cannot read all of %s\n", name);
				exit(1);
			}

			/*Sum R & L*/
			for (int i = 0; i < size; i++) {
				(*buf)[i] = temp_buf[i * 2] / 2.0 + temp_buf[i * 2 + 1] / 2.0;
			}

			delete[] temp_buf;

		}
		else {
			fprintf(stderr, "ERROR: %s : Only mono or stereo accepted", name);
			exit(1);
		}
	}
	return size;
}

void cudaFFT(int argc, char **argv, Data *p) {
	
	std::string input = "media/Castanets-441.wav";
	std::string reverb = "media/s1_r1_b_441_mono.wav";
	if (argc == 2) {
		if (argv[1][1] != '>')
			input = argv[1];
	}
	if (argc == 3) {
		input = argv[1];
		reverb = argv[2];
	}

	float *ibuf, *rbuf;
	int SIGNAL_SIZE = 0, FILTER_KERNEL_SIZE = 0;

	fprintf(stderr, "Reading input file\n");
	int inputCh;
	SIGNAL_SIZE = readFile(input.c_str(), &ibuf, inputCh);

	fprintf(stderr, "Reading reverb file\n");
	FILTER_KERNEL_SIZE = readFile(reverb.c_str(), &rbuf, inputCh);
	if (inputCh != 1) {
		fprintf(stderr, "ERROR: Only mono reverb sources accepted");
		exit(2);
	}

	findCudaDevice(argc, (const char **)argv);
	if (reverbFlag) {
		fprintf(stderr, "Doing GPU Convolution\n");
		/*Pad signal and filter kernel to same length*/
		float* h_padded_signal;
		float* h_padded_filter_kernel;
		//new_size = SIGNAL_SIZE + (FILTER_KERNEL_SIZE) % 2
		int new_size = PadData(ibuf, &h_padded_signal, SIGNAL_SIZE,
			rbuf, &h_padded_filter_kernel, FILTER_KERNEL_SIZE);
		int mem_size = sizeof(float) * new_size;


		/*MOVING SIGNAL TO GPU*/
		// Allocate device memory for signal
		float* d_signal;
		checkCudaErrors(hipMalloc((void**)&d_signal, mem_size));

		// Copy signal from host to device
		checkCudaErrors(hipMemcpy(d_signal, h_padded_signal, mem_size,
			hipMemcpyHostToDevice));

		/*MOVING IMPULSE TO GPU*/
		// Allocate device memory for filter kernel
		float* d_filter_kernel;
		checkCudaErrors(hipMalloc((void**)&d_filter_kernel, mem_size));
		// Copy host memory to device
		checkCudaErrors(hipMemcpy(d_filter_kernel, h_padded_filter_kernel, mem_size,
			hipMemcpyHostToDevice));

		/*FIND RMS OF ORIGINAL SIGNAL*/
		/*Convert raw float pointer into a thrust device pointer*/
		thrust::device_ptr<float> thrust_d_signal(d_signal);

		/*Declare thrust operators*/
		square<float> unary_op;
		thrust::plus<float> binary_op;

		/*Perform thrust reduction to find rms*/
		float rms = std::sqrt(thrust::transform_reduce(thrust_d_signal, thrust_d_signal + new_size, unary_op, 0.0f, binary_op) / new_size);

		///////////////////////////////////////////////////////////////////////////////
		/*GPU PROCESSING*/
		///////////////////////////////////////////////////////////////////////////////

		// CUFFT plan simple API
		hipfftHandle plan;
		CHECK_CUFFT_ERRORS(hipfftPlan1d(&plan, new_size, HIPFFT_R2C, 1));
		hipfftHandle outplan;
		CHECK_CUFFT_ERRORS(hipfftPlan1d(&outplan, new_size, HIPFFT_C2R, 1));

		/*Create complex arrays*/
		hipfftComplex* d_sig_complex;
		checkCudaErrors(hipMalloc(&d_sig_complex, new_size * sizeof(hipfftComplex)));
		hipfftComplex* d_filter_complex;
		checkCudaErrors(hipMalloc(&d_filter_complex, new_size * sizeof(hipfftComplex)));

		/*FFT*/
		printf("Transforming signal hipfftExecR2C\n");
		CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal*)d_signal, d_sig_complex));
		CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal*)d_filter_kernel, d_filter_complex));

		/*CONVOLUTION*/
		// Multiply the coefficients together and normalize the result
		printf("Launching ComplexPointwiseMulAndScale<<< >>>\n");
		int blockSize = 256;
		int numBlocks = (new_size + blockSize - 1) / blockSize;
		ComplexPointwiseMulAndScale << < numBlocks, blockSize >> > (d_sig_complex, d_filter_complex, new_size, 1.0f / new_size);
		// Check if kernel execution generated and error
		getLastCudaError("Kernel execution failed [ ComplexPointwiseMulAndScale ]");

		/*IFFT*/
		// Transform signal back
		printf("Transforming signal back hipfftExecC2R\n");
		CHECK_CUFFT_ERRORS(hipfftExecC2R(outplan, d_sig_complex, d_signal));

		if (hipDeviceSynchronize() != hipSuccess) {
			fprintf(stderr, "Cuda error: failed to synchronize\n");
		}

		/*Find RMS of resulting signal*/
		thrust::device_ptr<float> thrust_d_output_signal2(d_signal);
		float rms2 = std::sqrt(thrust::transform_reduce(thrust_d_signal, thrust_d_signal + new_size, unary_op, 0.0f, binary_op) / new_size);
		printf("RMS1: %f RMS2: %f\n", rms, rms2);

		/*Scale resulting signal according to input signal*/
		MyFloatScale << < numBlocks, blockSize >> > (d_signal, new_size, rms / rms2);

		/*MOVE BACK TO CPU & STORE IN STRUCT*/
		float* obuf = (float*)malloc(sizeof(float) * new_size);
		checkCudaErrors(hipMemcpy(obuf, d_signal, new_size * sizeof(float), hipMemcpyDeviceToHost));
		p->all_sources[0].buf = obuf;
		p->all_sources[0].length = new_size;

		fprintf(stderr, "Samples: %i\nTotal Bytes: %i\nTotal KB: %f3\nTotal MB: %f3\n\n\n", new_size, mem_size, mem_size / (float)1024, mem_size / (float)1024 / (float)1024);

		/*Write reverberated sound file*/
		//SndfileHandle file = SndfileHandle("output.wav", SFM_WRITE, isfinfo.format, isfinfo.channels, isfinfo.samplerate);
		//file.write(obuf, new_size);

		/*Destroy CUFFT context*/
		CHECK_CUFFT_ERRORS(hipfftDestroy(plan));
		CHECK_CUFFT_ERRORS(hipfftDestroy(outplan));

		/*Free memory*/

		free(ibuf);
		free(rbuf);

		free(h_padded_signal);
		free(h_padded_filter_kernel);

		checkCudaErrors(hipFree(d_signal));
		checkCudaErrors(hipFree(d_filter_kernel));
		checkCudaErrors(hipFree(d_sig_complex));
		checkCudaErrors(hipFree(d_filter_complex));
	}
	else {
		p->all_sources[0].buf = ibuf;
		p->all_sources[0].length = SIGNAL_SIZE;
		free(rbuf);
	}
	


}

void transform_hrtfs() {

	hipfftHandle plan;
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&plan, PAD_LEN, HIPFFT_R2C, NUM_HRTF * 2));
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, d_hrtf, (hipfftComplex*)d_hrtf));
	CHECK_CUFFT_ERRORS(hipfftDestroy(plan));

#ifdef CPU_FD_BASIC
	float max_diff = 0;
	hipfftComplex* buf = new hipfftComplex[NUM_HRTF * 2 * (PAD_LEN / 2 + 1)];
	checkCudaErrors(hipMemcpy(buf, d_hrtf, NUM_HRTF * 2 * (PAD_LEN / 2 + 1) * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
	int size = NUM_HRTF * HRTF_CHN * (PAD_LEN + 2);
	for (int i = 0; i < size; i++) {
		float gpu_val = *(((float*)buf) + i);
		float cpu_val = *(((float*)fft_hrtf) + i);
		float diff = fabs(gpu_val - cpu_val);
		if (max_diff < diff) {
			max_diff = diff;
		}
	}
	printf("GPU & CPU FFT are within %f precision\n", max_diff);
	delete[] buf;
#endif
}