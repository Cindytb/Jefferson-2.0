#include "hip/hip_runtime.h"
#include "cudaPart.cuh"
struct is_negative {
	__host__ __device__ bool operator()(float x) {
		if (x < 0) {
			return true;
		}
		return false;
	}
};
template <typename T>
struct square
{
	__host__ __device__
		T operator()(const T& x) const
	{
		return x * x;
	}
};
typedef float2 Complex; 
int readFile(const char *name, float **buf, int &numCh) {
	SF_INFO info;
	SNDFILE *sndfile;
	memset(&info, 0, sizeof(info));
	info.format = 0;
	sndfile = sf_open(name, SFM_READ, &info);
	if (sndfile == NULL) {
		fprintf(stderr, "ERROR. Cannot open %s\n", name);
		exit(1);
	}

	int size = info.frames;
	numCh = info.channels;

	*buf = (float*)malloc(sizeof(float) * size);

	if (info.channels == 1) {
		sf_read_float(sndfile, *buf, size);
	}

	else {
		/*Sum into mono & do RMS*/
		if (info.channels == 2) {
			/*Allocate temporary memory for wave file*/
			float *temp_buf = (float*)malloc(sizeof(float) * info.frames * 2);

			/*Read wave file into temporary memory*/
			sf_read_float(sndfile, temp_buf, info.frames * 2);

			/*Sum R & L*/
			for (int i = 0; i < info.frames; i++) {
				*buf[i] = temp_buf[i * 2] / 2.0 + temp_buf[i * 2 + 1] / 2.0;
			}

			free(temp_buf);

		}
		else {
			fprintf(stderr, "ERROR: %s : Only mono or stereo accepted", name);
		}
	}
	sf_close(sndfile);
	return size;
}

void cudaFFT(int argc, char **argv, Data *p) {

	std::string input = "media/Taiklatalvi.wav";
	std::string reverb = "media/medieval_church.wav";
	if (argc == 2) {
		if (argv[1][1] != '>')
			input = argv[1];
	}
	if (argc == 3) {
		input = argv[1];
		reverb = argv[2];
	}

	float *ibuf, *rbuf;
	int SIGNAL_SIZE = 0, FILTER_KERNEL_SIZE = 0;

	fprintf(stderr, "Reading input file\n");
	int inputCh;
	SIGNAL_SIZE = readFile(input.c_str(), &ibuf, inputCh);

	fprintf(stderr, "Reading reverb file\n");
	FILTER_KERNEL_SIZE = readFile(reverb.c_str(), &rbuf, inputCh);
	if (inputCh != 1) {
		fprintf(stderr, "ERROR: Only mono reverb sources accepted");
		exit(2);
	}

	findCudaDevice(argc, (const char **)argv);

	fprintf(stderr, "Doing GPU Convolution\n");
	/*Pad signal and filter kernel to same length*/
	float *h_padded_signal;
	float *h_padded_filter_kernel;
	//new_size = SIGNAL_SIZE + (FILTER_KERNEL_SIZE) % 2
	int new_size = PadData(ibuf, &h_padded_signal, SIGNAL_SIZE,
		rbuf, &h_padded_filter_kernel, FILTER_KERNEL_SIZE);
	int mem_size = sizeof(float) * new_size;


	/*MOVING SIGNAL TO GPU*/
	// Allocate device memory for signal
	float *d_signal;
	checkCudaErrors(hipMalloc((void **)&d_signal, mem_size));

	// Copy signal from host to device
	checkCudaErrors(hipMemcpy(d_signal, h_padded_signal, mem_size,
		hipMemcpyHostToDevice));

	/*MOVING IMPULSE TO GPU*/
	// Allocate device memory for filter kernel
	float *d_filter_kernel;
	checkCudaErrors(hipMalloc((void **)&d_filter_kernel, mem_size));
	// Copy host memory to device
	checkCudaErrors(hipMemcpy(d_filter_kernel, h_padded_filter_kernel, mem_size,
		hipMemcpyHostToDevice));

	/*FIND RMS OF ORIGINAL SIGNAL*/
	/*Convert raw float pointer into a thrust device pointer*/
	thrust::device_ptr<float> thrust_d_signal(d_signal);

	/*Declare thrust operators*/
	square<float> unary_op;
	thrust::plus<float> binary_op;

	/*Perform thrust reduction to find rms*/
	float rms = std::sqrt(thrust::transform_reduce(thrust_d_signal, thrust_d_signal + new_size, unary_op, 0.0f, binary_op) / new_size);

	///////////////////////////////////////////////////////////////////////////////
	/*GPU PROCESSING*/
	///////////////////////////////////////////////////////////////////////////////

	// CUFFT plan simple API
	hipfftHandle plan;
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&plan, new_size, HIPFFT_R2C, 1));
	hipfftHandle outplan;
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&outplan, new_size, HIPFFT_C2R, 1));

	/*Create complex arrays*/
	hipfftComplex *d_sig_complex;
	checkCudaErrors(hipMalloc(&d_sig_complex, new_size * sizeof(hipfftComplex)));
	hipfftComplex *d_filter_complex;
	checkCudaErrors(hipMalloc(&d_filter_complex, new_size * sizeof(hipfftComplex)));

	/*FFT*/
	printf("Transforming signal hipfftExecR2C\n");
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal *)d_signal, d_sig_complex));
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal *)d_filter_kernel, d_filter_complex));

	/*CONVOLUTION*/
	// Multiply the coefficients together and normalize the result
	printf("Launching ComplexPointwiseMulAndScale<<< >>>\n");
	int blockSize = 256;
	int numBlocks = (new_size + blockSize - 1) / blockSize;
	ComplexPointwiseMulAndScale << < numBlocks, blockSize >> > (d_sig_complex, d_filter_complex, new_size, 1.0f / new_size);
	// Check if kernel execution generated and error
	getLastCudaError("Kernel execution failed [ ComplexPointwiseMulAndScale ]");

	/*IFFT*/
	// Transform signal back
	printf("Transforming signal back hipfftExecC2R\n");
	CHECK_CUFFT_ERRORS(hipfftExecC2R(outplan, d_sig_complex, d_signal));

	if (hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "Cuda error: failed to synchronize\n");
	}

	/*Find RMS of resulting signal*/
	thrust::device_ptr<float> thrust_d_output_signal2(d_signal);
	float rms2 = std::sqrt(thrust::transform_reduce(thrust_d_signal, thrust_d_signal + new_size, unary_op, 0.0f, binary_op) / new_size);
	printf("RMS1: %f RMS2: %f\n", rms, rms2);

	/*Scale resulting signal according to input signal*/
	MyFloatScale << < numBlocks, blockSize >> > (d_signal, new_size, rms / rms2);

	/*MOVE BACK TO CPU & STORE IN STRUCT*/
	float *obuf = (float*)malloc(sizeof(float) * new_size);
	checkCudaErrors(hipMemcpy(obuf, d_signal, new_size * sizeof(float), hipMemcpyDeviceToHost));
	p->all_sources[0].buf = obuf;
	p->all_sources[0].length = new_size;

	fprintf(stderr, "Samples: %i\nTotal Bytes: %i\nTotal KB: %f3\nTotal MB: %f3\n\n\n", new_size, mem_size, mem_size / (float)1024, mem_size / (float)1024 / (float)1024);

	/*Write reverberated sound file*/
	//SndfileHandle file = SndfileHandle("output.wav", SFM_WRITE, isfinfo.format, isfinfo.channels, isfinfo.samplerate);
	//file.write(obuf, new_size);

	/*Destroy CUFFT context*/
	CHECK_CUFFT_ERRORS(hipfftDestroy(plan));
	CHECK_CUFFT_ERRORS(hipfftDestroy(outplan));

	/*Free memory*/

	free(ibuf);
	free(rbuf);

	free(h_padded_signal);
	free(h_padded_filter_kernel);

	checkCudaErrors(hipFree(d_signal));
	checkCudaErrors(hipFree(d_filter_kernel));
	checkCudaErrors(hipFree(d_sig_complex));
	checkCudaErrors(hipFree(d_filter_complex));


}

void transform_hrtfs() {

	hipfftHandle plan;

	int n = PAD_LEN;
	/*hipfftResult hipfftPlanMany(hipfftHandle *plan, int rank, int *n,
		int *inembed, int istride, int idist,
		int *onembed, int ostride, int odist,
		hipfftType type, int batch);*/
		/*stride = skip length. Ex 1 = every element, 2 = every other element*/
			/*use for interleaving???*/
		/*idist/odist is space between batches of transforms*/
			/*need to check if odist is in terms of complex numbers or floats*/
		/*inembed/onembed are for 2D/3D, num elements per dimension*/
	/*CHECK_CUFFT_ERRORS(
		hipfftPlanMany(&plan, 1, &n,
			&n, 1, n + 2,
			&n, 1, n / 2 + 1,
			HIPFFT_R2C, NUM_HRTF * 2)
	);
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal*)d_hrtf, (hipfftComplex*)d_hrtf));*/

	CHECK_CUFFT_ERRORS(hipfftPlan1d(&plan, PAD_LEN, HIPFFT_R2C, NUM_HRTF * 2));
	size_t size = sizeof(float) * NUM_HRTF * HRTF_CHN * (PAD_LEN + 2);
	float* d_dummy,* d_dummy2;
	checkCudaErrors(hipMalloc((void**)&d_dummy, size));
	checkCudaErrors(hipMalloc((void**)&d_dummy2, size));
	for (int i = 0; i < NUM_HRTF; i++) {
		hipfftReal* lbuf = d_dummy2 + i * HRTF_CHN * (PAD_LEN + 2);
		hipfftReal* rbuf = d_dummy2 + i * HRTF_CHN * (PAD_LEN + 2) + PAD_LEN + 2;
		CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal*) lbuf, (hipfftComplex*)lbuf));
		checkCudaErrors(hipDeviceSynchronize());
		CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal*) rbuf, (hipfftComplex*)rbuf));
		checkCudaErrors(hipDeviceSynchronize());
	}
	
	CHECK_CUFFT_ERRORS(hipfftDestroy(plan));
}