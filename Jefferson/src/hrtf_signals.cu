#include "hip/hip_runtime.h"
#include "hrtf_signals.cuh"

float* d_hrtf;
int elevation_pos[NUM_ELEV] =
{ -40,  -30,  -20,  -10,    0,   10,   20,   30,   40,   50,    60,    70,    80,  90 };
float azimuth_inc[NUM_ELEV] =
{ 6.43f, 6.00f, 5.00f, 5.00f, 5.00f, 5.00f, 5.00f, 6.00f, 6.43f, 8.00f, 10.00f, 15.00f, 30.00f, 361.0f };
//56	+ 60	+ 72 + 72	+ 72	+ 72  + 72		+ 60	+ 56 + 45 + 35		+ 24	+ 12	+ 1 = 709
int azimuth_offset[NUM_ELEV + 1];


/* on entry obj_ele and obj_azi are the new object position
* on exit hrtf_idx is set to the HRTF index of the closest HRTF position
*  hrtf_idx > 0 indicates to use right half-sphere HRTF
*  hrtf_idx < 0 indicates to create left half-sphere HRTF by exchanging L, R
*/
int pick_hrtf(float obj_ele, float obj_azi)
{
	int i, n, ele_idx, obj_azi_sign, hrtf_idx;
	float d, dmin;

	/* find closest elevation position */
	obj_ele = std::round(obj_ele / 10) * 10;
	dmin = 1e37f;
	for (i = 0; i < NUM_ELEV; i++) {
		d = obj_ele - elevation_pos[i];
		d = d > 0 ? d : -d;
		if (d < dmin) {
			dmin = d;
			ele_idx = i;
		}
	}
	/* find closest azimuth position */
	obj_azi = std::round(obj_azi);
	dmin = 1e37f;
	n = azimuth_offset[ele_idx + 1] - azimuth_offset[ele_idx];
	for (i = 0; i < n; i++) {
		d = obj_azi - i * azimuth_inc[ele_idx];
		d = d > 0 ? d : -d;
		if (d < dmin) {
			dmin = d;
			hrtf_idx = azimuth_offset[ele_idx] + i;
		}
	}

	/* return hrtf index */
	return(hrtf_idx);
}


/*HRTF Impulse reading for GPU/DRAM*/
int read_and_error_check(char* input, float* hrtf) {
	/* sndfile data structures */
	SNDFILE* sndfile;
	SF_INFO sfinfo;
	/* zero libsndfile structures */
	memset(&sfinfo, 0, sizeof(sfinfo));
	/* open hrtf file */
	if ((sndfile = sf_open(input, SFM_READ, &sfinfo)) == NULL) {
		fprintf(stderr, "Error: could not open hrtf file:\n%s\n", input);
		fprintf(stderr, "%s\n", sf_strerror(sndfile));
		return -1;
	}
	/* check signal parameters */
	if (sfinfo.channels != 1) {
		fprintf(stderr, "ERROR: incorrect number of channels in HRTF\n");
		return -1;
	}
	if (sfinfo.samplerate != SAMP_RATE) {
		fprintf(stderr, "ERROR: incorrect sampling rate\n");
		return -1;
	}
	/* read HRTF signal */
	unsigned num_samples = sfinfo.frames * sfinfo.channels;

	if (sf_read_float(sndfile, hrtf, num_samples) != num_samples) {
		fprintf(stderr, "ERROR: cannot read HRTF signal\n");
		return -1;
	}

	/* close file */
	sf_close(sndfile);

}
int read_hrtf_signals(void) {
	float* hrtf = new float[NUM_HRFT * HRTF_CHN * HRTF_LEN];   /* de-interleaved HRTF impulse responses */
	char hrtf_file[PATH_LEN];
	int i, j, ele, num_samples;
	float azi;


	j = 0;
	azimuth_offset[0] = 0;
	size_t size = sizeof(float) * NUM_HRFT * HRTF_LEN * HRTF_CHN;
	checkCudaErrors(hipMalloc((void**)&d_hrtf, size));
	for (i = 0; i < NUM_ELEV; i++) {
		ele = elevation_pos[i];
		for (azi = 0; azi < 360; azi += azimuth_inc[i]) {


			sprintf(hrtf_file, "%s/elev%d/L%de%03da.wav", HRTF_DIR, ele, ele, (int)round(azi));
			/* Print file information */
			printf("%3d %3d %s\n", i, j, hrtf_file);
			if (read_and_error_check(hrtf_file, hrtf + j * HRTF_CHN * HRTF_LEN)) {
				return -1;
			}

			sprintf(hrtf_file, "%s/elev%d/R%de%03da.wav", HRTF_DIR, ele, ele, (int)round(azi));
			printf("%3d %3d %s\n", i, j, hrtf_file);
			if (read_and_error_check(hrtf_file, hrtf + j * HRTF_CHN * HRTF_LEN + HRTF_LEN)) {
				return -1;
			}
			j++;
		}

		azimuth_offset[i + 1] = j;
	}
	checkCudaErrors(hipMemcpy(d_hrtf, hrtf, size, hipMemcpyHostToDevice));
	printf("\nHRTF index offsets for each elevation:\n");
	for (i = 0; i < NUM_ELEV + 1; i++) {
		printf("%3d ", azimuth_offset[i]);
	}
	printf("\n");
	delete[] hrtf;
	return 0;
}
////////////////////////////////////////////////////////////////////////////////



__global__ void timeDomainConvolutionNaive(float* ibuf, float* rbuf, float* obuf, long long oframes,
	long long rframes, int ch, float gain) {
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	float value = 0;
	for (int k = 0; k < rframes; k++) {
		value += ibuf[threadID - k] * rbuf[k];
	}
	obuf[threadID * 2 + ch] = value * gain;

}

////////////////////////////////////////////////////////////////////////////////
/*GPU Convolution was not fast enough because of the large overhead
of FFT and IFFT. Keeping the code here for future purposes*/
void GPUconvolve_hrtf(float* input, int hrtf_idx, float* d_output, int outputLen, float gain, hipStream_t* streams) {
	if (gain > 1)
		gain = 1;

	int numBlocks = 8;
	int numThread = FRAMES_PER_BUFFER / numBlocks;
	timeDomainConvolutionNaive << < numBlocks, numThread, 0, streams[0] >> > (
		input,
		d_hrtf + hrtf_idx * HRTF_CHN * HRTF_LEN,
		d_output,
		outputLen,
		HRTF_LEN,
		0,
		gain);
	timeDomainConvolutionNaive << < numBlocks, numThread, 0, streams[1] >> > (
		input,
		d_hrtf + hrtf_idx * HRTF_CHN * HRTF_LEN + HRTF_LEN,
		d_output,
		outputLen,
		HRTF_LEN,
		1,
		gain);

}
