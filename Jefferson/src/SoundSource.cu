#include "hip/hip_runtime.h"
﻿#include "SoundSource.cuh"

SoundSource::SoundSource() {
	count = 0;
	length = 0;
	gain = 0.99074;
	hrtf_idx = 314;
	coordinates.x = 1;
	coordinates.y = 0;
	coordinates.z = 0;
	azi = 270;
	ele = 0;
	streams = new hipStream_t[FLIGHT_NUM * STREAMS_PER_FLIGHT];
	for (int i = 0; i < FLIGHT_NUM; i++) {
		/*Allocating pinned memory for incoming transfer*/
		checkCudaErrors(hipHostMalloc(x + i, (PAD_LEN + 2) * sizeof(float)));
		/*Allocating memory for the inputs*/
		checkCudaErrors(hipMalloc(d_input + i, (PAD_LEN + 2) * sizeof(float)));
		/*Allocating memory for the outputs*/
		checkCudaErrors(hipMalloc(d_output + i, HRTF_CHN * (PAD_LEN + 2) * sizeof(float)));
		/*Allocating memory for the distance factor*/
		checkCudaErrors(hipMalloc(distance_factor + i, (PAD_LEN / 2 + 1) * sizeof(hipfftComplex)));
		/*Creating the streams*/
		for(int j = 0; j < STREAMS_PER_FLIGHT; j++){
			checkCudaErrors(hipStreamCreate(streams + i * STREAMS_PER_FLIGHT + j));
		}
		checkCudaErrors(hipMalloc(d_convbufs + i, 4 * HRTF_CHN * (PAD_LEN / 2 + 1) * sizeof(hipfftComplex)));

		/*Allocating pinned memory for outgoing transfer*/
		checkCudaErrors(hipHostMalloc(intermediate + i, (FRAMES_PER_BUFFER * HRTF_CHN) * sizeof(float)));
	}
	for (int i = 0; i < FLIGHT_NUM; i++) {
		for (int j = 0; j < PAD_LEN + 2; j++) {
			x[i][j] = 0.0f;
		}
	}
	fftw_intermediate = fftwf_alloc_complex(2 * (PAD_LEN / 2 + 1));
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&in_plan, PAD_LEN, HIPFFT_R2C, 1));
	/*hipfftResult hipfftPlanMany(hipfftHandle *plan, int rank, int *n,
		int *inembed, int istride, int idist,
		int *onembed, int ostride, int odist,
		hipfftType type, int batch);*/
		/*stride = skip length for interleaving. Ex 1 = every element, 2 = every other element*/
			/*use for interleaving*/
		/*idist/odist is space between batches of transforms*/
			/*need to check if odist is in terms of complex numbers or floats*/
		/*inembed/onembed are for 2D/3D, num elements per dimension*/
	/*This type of cufft plan will take 2 mono channels located contiguously in memory, take the IFFT, and interleave them*/
	int n = PAD_LEN;
	CHECK_CUFFT_ERRORS(
		hipfftPlanMany(
			&out_plan, 1, &n,
			&n, 1, n / 2 + 1,
			&n, 2, 1,
			HIPFFT_C2R, 2)
	);
	fftw_in_plan = fftwf_plan_dft_r2c_1d(PAD_LEN, x[0], fftw_intermediate, FFTW_ESTIMATE);
	fftw_out_plan = fftwf_plan_many_dft_c2r(
		1, &PAD_LEN, 2, 
		fftw_intermediate, NULL, 
		1, PAD_LEN / 2 + 1, 
		(float*)fftw_intermediate, NULL, 
		2, 1, FFTW_ESTIMATE
	);

}
void SoundSource::updateInfo() {
	/*Calculate the radius, distance, elevation, and azimuth*/
	float r = std::sqrt(coordinates.x * coordinates.x + coordinates.z * coordinates.z + coordinates.y * coordinates.y);
	float horizR = std::sqrt(coordinates.x * coordinates.x + coordinates.z * coordinates.z);
	ele = (float)atan2(coordinates.y, horizR) * 180.0f / PI;

	azi = atan2(-coordinates.x / r, coordinates.z / r) * 180.0f / PI;
	if (azi < 0.0f) {
		azi += 360;
	}
	ele = round(ele);
	azi = round(azi);
	hrtf_idx = pick_hrtf(ele, azi);
}

void SoundSource::drawWaveform() {
	float rotateVBO_y = atan2(-coordinates.z, coordinates.x) * 180.0f / PI;

	if (rotateVBO_y < 0) {
		rotateVBO_y += 360;
	}
	waveform->averageNum = 100;
	waveform->update();
	waveform->draw(rotateVBO_y, ele, 0.0f);
}
void SoundSource::interpolationCalculations(int* hrtf_indices, float* omegas) {
	float omegaA, omegaB, omegaC, omegaD, omegaE, omegaF;
	int phi[2];
	int theta[4];
	float deltaTheta1, deltaTheta2;
	phi[0] = int(ele) / 10 * 10; /*hard coded 10 because the elevation increments in the KEMAR HRTF database is 10 degrees*/
	phi[1] = int(ele + 9) / 10 * 10;
	omegaE = (ele - phi[0]) / 10.0f;
	omegaF = (phi[1] - ele) / 10.0f;

	for (int i = 0; i < NUM_ELEV; i++) {
		if (phi[0] == elevation_pos[i]) {
			deltaTheta1 = azimuth_inc[i];
		}
		if (phi[1] == elevation_pos[i]) {
			deltaTheta2 = azimuth_inc[i];
			break;
		}
	}
	theta[0] = int(azi / deltaTheta1) * deltaTheta1;
	theta[1] = int((azi + deltaTheta1 - 1) / deltaTheta1) * deltaTheta1;
	theta[2] = int(azi / deltaTheta2) * deltaTheta2;
	theta[3] = int((azi + deltaTheta2 - 1) / deltaTheta2) * deltaTheta2;
	omegaA = (azi - theta[0]) / deltaTheta1;
	omegaB = (theta[1] - azi) / deltaTheta1;
	omegaC = (azi - theta[2]) / deltaTheta2;
	omegaD = (theta[3] - azi) / deltaTheta2;

	hrtf_indices[0] = pick_hrtf(phi[0], theta[0]);
	hrtf_indices[1] = pick_hrtf(phi[0], theta[1]);
	hrtf_indices[2] = pick_hrtf(phi[1], theta[2]);
	hrtf_indices[3] = pick_hrtf(phi[1], theta[3]);

	omegas[0] = omegaA;
	omegas[1] = omegaB;
	omegas[2] = omegaC;
	omegas[3] = omegaD;
	omegas[4] = omegaE;
	omegas[5] = omegaF;

}
/*
	R(r) = (1 / (1 + (fs / vs) (r - r0)^2) ) * e^ ((-j2PI (fs/vs) * (r - r0) *k) / N)
			|----------FRAC-----------------|	  |------------exponent------------------|

	FRAC * e^(exponent)
	FRAC * (cosine(exponent) - sine(exponent))
	R[r].x = FRAC * cosine(exponent)
	R[r].y = -FRAC * sine(exponent)
	*/
void SoundSource::calculateDistanceFactor(int blockNo){
	hipfftComplex* d_distance_factor = this->distance_factor[blockNo % FLIGHT_NUM];
	hipStream_t* streams = this->streams + (blockNo * 2 % FLIGHT_NUM);
	float r = std::sqrt(
		coordinates.x * coordinates.x + 
		coordinates.y * coordinates.y + 
		coordinates.z * coordinates.z
	);
	r /= 5;
	float fsvs = 44100.0 / 343.0;
	float frac = 1 + fsvs * pow(r, 2);
	float N = PAD_LEN / 2 + 1;
	int numThreads = 256;
	int numBlocks = (PAD_LEN / 2 + numThreads ) / numThreads;
	generateDistanceFactor << < numThreads, numBlocks, 0, streams[1] >> > (d_distance_factor, frac, fsvs, r, N);

}
/*
This method is a slightly tweaked implementation of Jose Belloch's
"Headphone-Based Virtual Spatialization of Sound with a GPU Accelerator"
paper from the Journal of the Audio Engineering Society,
Volume 61, No 7/8, 2013, July/August
*/
void SoundSource::allKernels(float* d_input, float* d_output, 
	hipfftComplex* d_convbufs, hipfftComplex* d_distance_factor, 
	hipStream_t* streams, float* omegas, int* hrtf_indices){
	fillWithZeroes(&d_output, 0, 2 * (PAD_LEN + 2));

	CHECK_CUFFT_ERRORS(hipfftSetStream(in_plan, streams[0]));
	CHECK_CUFFT_ERRORS(hipfftSetStream(out_plan, streams[0]));
	float scale = 1.0f / ((float)PAD_LEN);
	CHECK_CUFFT_ERRORS(hipfftExecR2C(in_plan, (hipfftReal*)d_input, (hipfftComplex*)d_input));
	checkCudaErrors(hipStreamSynchronize(streams[0]));
	int numThreads = 256;
	int numBlocks = (PAD_LEN / 2 + numThreads) / numThreads;
	size_t buf_size = PAD_LEN + 2;
	/*The azi & ele falls exactly on an hrtf resolution*/
	if (hrtf_indices[0] == hrtf_indices[1] && hrtf_indices[1] == hrtf_indices[2] && hrtf_indices[2] == hrtf_indices[3]) {
		/*+ Theta 1 Left*/
		ComplexPointwiseMulAndScaleOutPlace << < numBlocks, numThreads, 0, streams[0] >> > (
			(hipfftComplex*)d_input,
			(hipfftComplex*)(d_hrtf + hrtf_indices[0] * (PAD_LEN + 2) * HRTF_CHN),
			d_convbufs,
			PAD_LEN / 2 + 1,
			scale
			);
		ComplexPointwiseMulInPlace << < numBlocks, numThreads, 0, streams[0] >> > (
			d_distance_factor, 
			d_convbufs, 
			PAD_LEN / 2 + 1
			);
		ComplexPointwiseAdd << < numBlocks, numThreads, 0, streams[0] >> > (
			d_convbufs,
			(hipfftComplex*)d_output,
			PAD_LEN / 2 + 1
			);
		/*+ Theta 1 Right*/
		ComplexPointwiseMulAndScaleOutPlace << < numBlocks, numThreads, 0, streams[1] >> > (
			(hipfftComplex*)d_input,
			(hipfftComplex*)(d_hrtf + hrtf_indices[0] * (PAD_LEN + 2) * HRTF_CHN + PAD_LEN + 2),
			d_convbufs + buf_size / 2,
			PAD_LEN / 2 + 1,
			scale
			);
		ComplexPointwiseMulInPlace << < numBlocks, numThreads, 0, streams[1] >> > (
			d_distance_factor, 
			d_convbufs + buf_size / 2, 
			PAD_LEN / 2 + 1
			);
		
		ComplexPointwiseAdd << < numBlocks, numThreads, 0, streams[1] >> > (
			d_convbufs + buf_size / 2,
			(hipfftComplex*)(d_output + buf_size),
			PAD_LEN / 2 + 1
			);
	}
	/*If the elevation falls on the resolution, interpolate the azimuth*/
	else if (hrtf_indices[0] == hrtf_indices[2]) {
		for (int buf_no = 0; buf_no < 4; buf_no++) {
			/*Even buf numbers are the left channel, odd ones are the right channel*/
			float curr_scale;
			if (buf_no < 2)
				curr_scale = scale * omegas[1];
			else {
				curr_scale = scale * omegas[0];
			}
			int hrtf_index;
			if (buf_no < 2)
				hrtf_index = hrtf_indices[0];
			else
				hrtf_index = hrtf_indices[1];
			ComplexPointwiseMulAndScaleOutPlace << < numBlocks, numThreads, 0, streams[buf_no] >> > (
				(hipfftComplex*)d_input,
				(hipfftComplex*)(d_hrtf + hrtf_index * (PAD_LEN + 2) * HRTF_CHN + ((buf_no % 2) * (PAD_LEN + 2))),
				d_convbufs + buf_size / 2 * buf_no,
				PAD_LEN / 2 + 1,
				curr_scale
				);
			ComplexPointwiseMulInPlace << < numBlocks, numThreads, 0, streams[buf_no] >> > (
				d_distance_factor, 
				d_convbufs + buf_size / 2 * buf_no,
				PAD_LEN / 2 + 1
				);
			ComplexPointwiseAdd << < numBlocks, numThreads, 0, streams[buf_no] >> > (
				d_convbufs + buf_size / 2 * buf_no,
				(hipfftComplex*)(d_output + buf_size * (buf_no % 2)),
				PAD_LEN / 2 + 1
				);
		}

	}
	/*If the azimuth falls on the resolution, interpolate the elevation*/
	else if (hrtf_indices[0] == hrtf_indices[1] && hrtf_indices[0] != hrtf_indices[2]) {
		for (int buf_no = 0; buf_no < 4; buf_no++) {
			/*Even buf numbers are the left channel, odd ones are the right channel*/
			float curr_scale;
			int hrtf_index;
			switch (buf_no) {
			case 0:
			case 1:
				curr_scale = scale * omegas[4];
				hrtf_index = 0;
				break;
			case 2:
			case 3:
				curr_scale = scale * omegas[5];
				hrtf_index = 2;
				break;
			}

			ComplexPointwiseMulAndScaleOutPlace << < numBlocks, numThreads, 0, streams[buf_no] >> > (
				(hipfftComplex*)d_input,
				(hipfftComplex*)(d_hrtf + hrtf_indices[hrtf_index] * (PAD_LEN + 2) * HRTF_CHN + ((buf_no % 2) * (PAD_LEN + 2))),
				d_convbufs + buf_size / 2 * buf_no,
				PAD_LEN / 2 + 1,
				curr_scale
				);
			ComplexPointwiseMulInPlace << < numBlocks, numThreads, 0, streams[buf_no] >> > (
				d_distance_factor,
				d_convbufs + buf_size / 2 * buf_no,
				PAD_LEN / 2 + 1
				);
			ComplexPointwiseAdd << < numBlocks, numThreads, 0, streams[buf_no] >> > (
				d_convbufs + buf_size / 2 * buf_no,
				(hipfftComplex*)(d_output + buf_size * (buf_no % 2)),
				PAD_LEN / 2 + 1
				);
		}
	}
	/*Worst case scenario*/
	else {
		for (int buf_no = 0; buf_no < 8; buf_no++) {
			/*Even buf numbers are the left channel, odd ones are the right channel*/
			float curr_scale;
			int hrtf_index = buf_no / 2;
			switch (hrtf_index) {
			case 0:
				curr_scale = scale * omegas[5] * omegas[1];
				break;
			case 1:
				curr_scale = scale * omegas[5] * omegas[0];
				break;
			case 2:
				curr_scale = scale * omegas[4] * omegas[3];
				break;
			case 3:
				curr_scale = scale * omegas[4] * omegas[2];
				break;
			}
			ComplexPointwiseMulAndScaleOutPlace << < numBlocks, numThreads, 0, streams[buf_no] >> > (
				(hipfftComplex*)d_input,
				(hipfftComplex*)(d_hrtf + hrtf_indices[hrtf_index] * (PAD_LEN + 2) * HRTF_CHN + ((buf_no % 2) * (PAD_LEN + 2))),
				d_convbufs + buf_size / 2 * buf_no,
				PAD_LEN / 2 + 1,
				curr_scale
				);
			ComplexPointwiseMulInPlace << < numBlocks, numThreads, 0, streams[buf_no] >> > (
				d_distance_factor,
				d_convbufs + buf_size / 2 * buf_no,
				PAD_LEN / 2 + 1
				);
			ComplexPointwiseAdd << < numBlocks, numThreads, 0, streams[buf_no] >> > (
				d_convbufs + buf_size / 2 * buf_no,
				(hipfftComplex*)(d_output + buf_size * (buf_no % 2)),
				PAD_LEN / 2 + 1
				);
		}
	}
}

void SoundSource::interpolateConvolve(int blockNo) {
	int hrtf_indices[4];
	float omegas[6];
	interpolationCalculations(hrtf_indices, omegas);
	calculateDistanceFactor(blockNo % FLIGHT_NUM);
	
	hipfftComplex* d_distance_factor = this->distance_factor[blockNo % FLIGHT_NUM];
	float* d_input = this->d_input[blockNo % FLIGHT_NUM];
	float* d_output = this->d_output[blockNo % FLIGHT_NUM];
	hipfftComplex* d_convbufs = this ->d_convbufs[blockNo % FLIGHT_NUM];
	hipStream_t* streams = this->streams + (blockNo * 2 % FLIGHT_NUM);

	allKernels(d_input, d_output, d_convbufs, d_distance_factor, streams, omegas, hrtf_indices);
	
	for (int i = 1; i < STREAMS_PER_FLIGHT; i++) {
		checkCudaErrors(hipStreamSynchronize(streams[i]));
	}
	CHECK_CUFFT_ERRORS(hipfftExecC2R(out_plan, (hipfftComplex*)d_output, d_output));
}
void SoundSource::fftConvolve(int blockNo) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	float* d_input = this->d_input[blockNo % FLIGHT_NUM];
	float* d_output = this->d_output[blockNo % FLIGHT_NUM];
	hipStream_t* streams = this->streams + (blockNo * 2 % FLIGHT_NUM);
	if (gain > 1)
		gain = 1;
	float scale = 1.0f / ((float) PAD_LEN);
	CHECK_CUFFT_ERRORS(hipfftExecR2C(in_plan, (hipfftReal*)d_input, (hipfftComplex*)d_input));
	int numThreads = 256;
	int numBlocks = (PAD_LEN / 2 + numThreads - 1) / numThreads;
	//interpolateConvolve(blockNo);
	ComplexPointwiseMulAndScaleOutPlace << < numBlocks, numThreads, 0, streams[0] >> > (
		(hipfftComplex*)d_input,
		(hipfftComplex*)(d_hrtf + hrtf_idx * (PAD_LEN + 2) * HRTF_CHN),
		(hipfftComplex*)d_output,
		PAD_LEN / 2 + 1,
		scale
	);

	ComplexPointwiseMulAndScaleOutPlace << < numBlocks, numThreads, 0, streams[1] >> > (
		(hipfftComplex*)d_input,
		(hipfftComplex*)(d_hrtf + hrtf_idx * (PAD_LEN + 2) * HRTF_CHN + PAD_LEN + 2),
		(hipfftComplex*)(d_output + PAD_LEN + 2),
		PAD_LEN / 2 + 1,
		scale
	);
	checkCudaErrors(hipStreamSynchronize(streams[0]));
	checkCudaErrors(hipStreamSynchronize(streams[1]));
	CHECK_CUFFT_ERRORS(hipfftExecC2R(out_plan, (hipfftComplex*)d_output, d_output));

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	sum_ms += milliseconds;
	num_iterations++;
	avg_ms = sum_ms / float(num_iterations);
	fprintf(stderr, "Average GPU Basic FD Kernel Time: %f\n", avg_ms);

}
/* convolve signal buffer with HRTF
* new signal starts at HRTF_LEN frames into x buffer
* x is mono input signal
* HRTF and y are interleaved by channel
* y_len is in frames
*/
void SoundSource::cpuTDConvolve(float *input, float *output, int outputLen, float gain){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	float *l_hrtf = hrtf + hrtf_idx * HRTF_CHN * (PAD_LEN + 2);
	float *r_hrtf = hrtf + hrtf_idx * HRTF_CHN * (PAD_LEN + 2) + PAD_LEN + 2;
	if (gain > 1)
		gain = 1;

	/* zero output buffer */
	for (int i = 0; i < outputLen * HRTF_CHN; i++) {
		output[i] = 0.0;
	}
	for (int n = 0; n < outputLen; n++) {
		for (int k = 0; k < HRTF_LEN; k++) {
			for (int j = 0; j < HRTF_CHN; j++) {
				/* outputLen and HRTF_LEN are n frames, output and hrtf are interleaved
				* input is mono
				*/
				if(j == 0){
					output[2 * n + j] += input[n - k] * l_hrtf[k];
				}
				else{
					output[2 * n + j] += input[n - k] * r_hrtf[k];
				}
				
			}
			output[2 * n] *= gain;
			output[2 * n + 1] *= gain;
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	sum_ms += milliseconds;
	num_iterations++;
	avg_ms = sum_ms / float(num_iterations);
	fprintf(stderr, "Average CPU Time Domain Kernel Time: %f\n", avg_ms);
}
void pointwiseMultiplication(fftwf_complex* a, fftwf_complex* b, int size) {
	for (int i = 0; i < size; i++) {
		fftwf_complex temp;
		temp[0] = a[i][0];
		temp[1] = a[i][1];
		a[i][0] = temp[0] * b[i][0] - temp[1] * b[i][1];
		a[i][1] = temp[0] * b[i][1] + temp[1] * b[i][0];
	}
}
void complexScaling(fftwf_complex* f_x, float scale, int size) {
	for (int i = 0; i < size; i++) {
		f_x[i][0] *= scale;
		f_x[i][1] *= scale;
	}
}
void SoundSource::cpuFFTConvolve() {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	float* output = intermediate[0];
	fftwf_execute(fftw_in_plan); /*FFT on x[0] --> fftw_intermediate*/
	complexScaling(fftw_intermediate, 1.0 / PAD_LEN, PAD_LEN / 2 + 1);
	/*Copying over for both channels*/
#pragma omp for parallel
	for (int i = 0; i < PAD_LEN / 2 + 1; i++) {
		fftw_intermediate[i + PAD_LEN / 2 + 1][0] = fftw_intermediate[i][0];
		fftw_intermediate[i + PAD_LEN / 2 + 1][1] = fftw_intermediate[i][1];
	}
	/*Doing both channels at once since they're contiguous in memory*/
	pointwiseMultiplication(fftw_intermediate, 
		fft_hrtf + hrtf_idx * HRTF_CHN * (PAD_LEN / 2 + 1), 
		PAD_LEN + 2);
	fftwf_execute(fftw_out_plan);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	sum_ms += milliseconds;
	num_iterations++;
	avg_ms = sum_ms / float(num_iterations);
	fprintf(stderr, "Average CPU Basic FD Kernel Time: %f\n", avg_ms);

}

////////////////////////////////////////////////////////////////////////////////
/*GPU Convolution was not fast enough because of the large overhead
of FFT and IFFT. Keeping the code here for future purposes*/
void SoundSource::gpuTDConvolve(float* input, float* d_output, int outputLen, float gain, hipStream_t* streams) {
	if (gain > 1)
		gain = 1;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	int numThread = 256;
	int numBlocks = (outputLen + numThread - 1) / numThread;
	timeDomainConvolutionNaive << < numBlocks, numThread, 0, streams[0] >> > (
		input,
		d_hrtf + hrtf_idx * HRTF_CHN * (PAD_LEN + 2),
		d_output,
		outputLen,
		HRTF_LEN,
		0,
		gain);
	timeDomainConvolutionNaive << < numBlocks, numThread, 0, streams[1] >> > (
		input,
		d_hrtf + hrtf_idx * HRTF_CHN * (PAD_LEN + 2) + PAD_LEN + 2,
		d_output,
		outputLen,
		HRTF_LEN,
		1,
		gain);
	checkCudaErrors(hipDeviceSynchronize());
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	sum_ms += milliseconds;
	num_iterations++;
	avg_ms = sum_ms / float(num_iterations);
	fprintf(stderr, "Average GPU Time Domain Kernel Time: %f\n", avg_ms);

}
void SoundSource::process(int blockNo){
#ifdef RT_GPU_INTERPOLATE
	 interpolateConvolve(blockNo);
#endif
#ifdef RT_GPU_BASIC
	fftConvolve(blockNo);
#endif
#ifdef RT_GPU_TD
	/*Process*/
	gpuTDConvolve(
		d_input[blockNo % FLIGHT_NUM] + PAD_LEN - FRAMES_PER_BUFFER,
		d_output[blockNo % FLIGHT_NUM] + 2 * (PAD_LEN - FRAMES_PER_BUFFER),
		FRAMES_PER_BUFFER,
		gain, streams + blockNo * STREAMS_PER_FLIGHT);

#endif
#ifdef CPU_FD_BASIC
	 cpuFFTConvolve();
#endif
}
void SoundSource::~SoundSource() {
	free(buf);
	CHECK_CUFFT_ERRORS(hipfftDestroy(in_plan));
	CHECK_CUFFT_ERRORS(hipfftDestroy(out_plan));

	for (int i = 0; i < FLIGHT_NUM; i++) {
		checkCudaErrors(hipHostFree(x[i]));
		checkCudaErrors(hipFree(d_input[i]));
		checkCudaErrors(hipFree(d_output[i]));
		checkCudaErrors(hipFree(distance_factor[i]));
		for (int j = 0; j < STREAMS_PER_FLIGHT; j++) {
			checkCudaErrors(hipStreamDestroy(streams[i * STREAMS_PER_FLIGHT + j]));
		}
		checkCudaErrors(hipFree(d_convbufs[i]));
		checkCudaErrors(hipHostFree(intermediate[i]));
	}
	free(streams);
	fftwf_free(fftw_intermediate);
}
