#include "hip/hip_runtime.h"
#include "SoundSource.cuh"

SoundSource::SoundSource() {
	count = 0;
	length = 0;
	gain = 0.99074;
	hrtf_idx = 314;
	azi = 270;
	ele = 0;
	streams = new hipStream_t[FLIGHT_NUM * 2];
	for (int i = 0; i < FLIGHT_NUM; i++) {
		/*Allocating pinned memory for incoming transfer*/
		checkCudaErrors(hipHostMalloc(x + i, (PAD_LEN + 2) * sizeof(float)));
		/*Allocating memory for the inputs*/
		checkCudaErrors(hipMalloc(d_input + i, (PAD_LEN + 2) * sizeof(float)));
		/*Allocating memory for the outputs*/
		checkCudaErrors(hipMalloc(d_output + i, HRTF_CHN * (PAD_LEN + 2) * sizeof(float)));
		/*Creating the streams*/
		checkCudaErrors(hipStreamCreate(streams + i * 2));
		checkCudaErrors(hipStreamCreate(streams + i * 2 + 1));
		/*Allocating pinned memory for outgoing transfer*/
		checkCudaErrors(hipHostMalloc(intermediate + i, (FRAMES_PER_BUFFER * HRTF_CHN) * sizeof(float)));
	}
	for (int i = 0; i < FLIGHT_NUM; i++) {
		for (int j = 0; j < PAD_LEN + 2; j++) {
			x[i][j] = 0.0f;
		}
	}
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&in_plan, PAD_LEN, HIPFFT_R2C, 1));
	/*hipfftResult hipfftPlanMany(hipfftHandle *plan, int rank, int *n,
		int *inembed, int istride, int idist,
		int *onembed, int ostride, int odist,
		hipfftType type, int batch);*/
		/*stride = skip length for interleaving. Ex 1 = every element, 2 = every other element*/
			/*use for interleaving*/
		/*idist/odist is space between batches of transforms*/
			/*need to check if odist is in terms of complex numbers or floats*/
		/*inembed/onembed are for 2D/3D, num elements per dimension*/
	/*This type of cufft plan will take 2 mono channels located contiguously in memory, take the IFFT, and interleave them*/
	int n = PAD_LEN;
	CHECK_CUFFT_ERRORS(
		hipfftPlanMany(
			&out_plan, 1, &n,
			&n, 1, n / 2 + 1,
			&n, 2, 1,
			HIPFFT_C2R, 2)
	);
}
void SoundSource::updateInfo() {
	/*Calculate the radius, distance, elevation, and azimuth*/
	float r = std::sqrt(coordinates.x * coordinates.x + coordinates.z * coordinates.z + coordinates.y * coordinates.y);
	float horizR = std::sqrt(coordinates.x * coordinates.x + coordinates.z * coordinates.z);
	ele = (float)atan2(coordinates.y, horizR) * 180.0f / PI;

	azi = atan2(-coordinates.x / r, coordinates.z / r) * 180.0f / PI;
	if (azi < 0.0f) {
		azi += 360;
	}
	float newR = r / 100 + 1;
	hrtf_idx = pick_hrtf(ele, azi);
}

void SoundSource::drawWaveform() {
	float rotateVBO_y = atan2(-coordinates.z, coordinates.x) * 180.0f / PI;

	if (rotateVBO_y < 0) {
		rotateVBO_y += 360;
	}
	waveform->averageNum = 100;
	waveform->update();
	waveform->draw(rotateVBO_y, ele, 0.0f);
}

void SoundSource::fftConvolve(int blockNo) {
	float* d_input = this->d_input[blockNo % FLIGHT_NUM];
	float* d_output = this->d_output[blockNo % FLIGHT_NUM];
	hipStream_t* streams = this->streams + (blockNo * 2 % FLIGHT_NUM);
	if (gain > 1)
		gain = 1;
	float scale = 1.0f / ((float) PAD_LEN);
	CHECK_CUFFT_ERRORS(hipfftExecR2C(in_plan, (hipfftReal*)d_input, (hipfftComplex*)d_input));
	int numThreads = 128;
	int numBlocks = (PAD_LEN + numThreads - 1) / numThreads;
	//interpolateConvolve(blockNo);
	ComplexPointwiseMulAndScaleOutPlace << < numBlocks, numThreads, 0, streams[0] >> > (
		(hipfftComplex*)d_input,
		(hipfftComplex*)(d_hrtf + hrtf_idx * (PAD_LEN + 2) * HRTF_CHN),
		(hipfftComplex*)d_output,
		PAD_LEN / 2 + 1,
		scale
	);

	ComplexPointwiseMulAndScaleOutPlace << < numBlocks, numThreads, 0, streams[1] >> > (
		(hipfftComplex*)d_input,
		(hipfftComplex*)(d_hrtf + hrtf_idx * (PAD_LEN + 2) * HRTF_CHN + PAD_LEN + 2),
		(hipfftComplex*)(d_output + PAD_LEN + 2),
		PAD_LEN / 2 + 1,
		scale
	);
	checkCudaErrors(hipStreamSynchronize(streams[0]));
	checkCudaErrors(hipStreamSynchronize(streams[1]));
	CHECK_CUFFT_ERRORS(hipfftExecC2R(out_plan, (hipfftComplex*)d_output, d_output));

}

SoundSource::~SoundSource() {
	CHECK_CUFFT_ERRORS(hipfftDestroy(in_plan));
	CHECK_CUFFT_ERRORS(hipfftDestroy(out_plan));
	for (int i = 0; i < FLIGHT_NUM; i++) {
		hipHostFree(x[i]);
		hipFree(d_input[i]);
		hipFree(d_output[i]);
		hipStreamDestroy(streams[i * 2]);
		hipStreamDestroy(streams[i * 2 + 1]);
		hipHostFree(intermediate[i]);
	}
}