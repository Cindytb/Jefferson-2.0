#include "hip/hip_runtime.h"
#include "vbo.cuh"

VBO::VBO(float **a, float *b, unsigned int c, float d)
	: d_buf(a), translate_x(b), numSamples(c), ratio(d)
{}
void VBO::init() {
	assert(&vbo);

	// create buffer object
	glGenBuffers(1, &vbo);
	glBindBuffer(GL_ARRAY_BUFFER, vbo);
	vboSize = numSamples * sizeof(float4);
	// initialize buffer object
	glBufferData(GL_ARRAY_BUFFER, vboSize, 0, GL_DYNAMIC_DRAW);

	glBindBuffer(GL_ARRAY_BUFFER, 0);

	// register this buffer object with CUDA
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo, cudaGraphicsMapFlagsWriteDiscard));

	SDK_CHECK_ERROR_GL();
}
void VBO::create() {
	// map OpenGL buffer object for writing from CUDA
	float4 *dptr;
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_vbo_resource, 0));
	size_t num_bytes;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
		cuda_vbo_resource));
	printf("CUDA mapped VBO: May access %zu bytes\n", num_bytes);

	fprintf(stderr, "Launching Kernel: number of samples: %d\n", numSamples);
	launch_new_kernel(dptr, *d_buf, numSamples, averageNum, ratio);

	// unmap buffer object
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));

}
void VBO::update() {
	// map OpenGL buffer object for writing from CUDA
	float4 *dptr;
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_vbo_resource, 0));
	size_t num_bytes;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
		cuda_vbo_resource));

	launch_new_kernel(dptr, *d_buf, numSamples, averageNum, ratio);

	// unmap buffer object
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
}
void VBO::draw(float rotateVBO_x, float rotateVBO_y, float rotateVBO_z) {
	glPushMatrix();

	// render from the vbo
	glRotatef(rotateVBO_x, 0.0, 1.0, 0);
	glRotatef(rotateVBO_y, 0, 0.0, 1.0);
	//glRotatef(rotateVBO_z, 0, 0, 1.0);
	glTranslatef(*translate_x, 0.0, 0.0);
	glBindBuffer(GL_ARRAY_BUFFER, vbo);
	glVertexPointer(4, GL_FLOAT, 0, 0);

	glEnableClientState(GL_VERTEX_ARRAY);
	glColor3f(1.0, 1.0, 1.0);
	//glDrawArrays(GL_POINTS, 0, vboSize);
	glLineWidth(0.005f);
	glEnable(GL_LINE_SMOOTH);
	glDrawArrays(GL_LINES, 0, vboSize);
	glDisableClientState(GL_VERTEX_ARRAY);

	glPopMatrix();
}
VBO::~VBO() {
	// unregister this buffer object with CUDA
	checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));

	glBindBuffer(1, vbo);
	glDeleteBuffers(1, &vbo);
	hipFree(&d_buf);
	fprintf(stderr, "Freed the device audio buffer\n");
}

void launch_new_kernel(float4 *pos, float* buf, unsigned int size, int averageNum, float ratio) {
	unsigned const int numThreads = 1024;
	int numBlocks = size / numThreads + 1;

	int averageSize = averageNum;
	int reducedSize;
	if (averageSize < 1) {
		reducedSize = size;
	}
	else {

		reducedSize = size / averageSize;
	}
	numBlocks = reducedSize / numThreads + 1;
	averagingKernel << < numBlocks, numThreads >> > (pos, buf, size, ratio, averageSize);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}