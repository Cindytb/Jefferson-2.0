#include "main.cuh"
#include <chrono>
#include <thread>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
int main(int argc, char *argv[]){
	if (argc > 3 ) {
		fprintf(stderr, "Usage: %s input.wav reverb.wav", argv[0]);
		return 0;
	}
	
	data.count = 0;
	data.length = 0;
	data.gain = 0.99074;
	data.hrtf_idx = 151;
	#if(DEBUGMODE != 1)
		/*Initialize & read files*/
		cudaFFT(argc, argv, p);
	
		fprintf(stderr, "Opening and Reading HRTF signals\n");
		/*Open & read hrtf files*/

		if (read_hrtf_signals() != 0) {
			exit(EXIT_FAILURE);
		}
		p->hrtf_idx = 0;


		fprintf(stderr, "Opening output file\n");
		SF_INFO osfinfo;
		osfinfo.channels = 2;
		osfinfo.samplerate = 44100;
		osfinfo.format = SF_FORMAT_PCM_24 | SF_FORMAT_WAV;
		p->sndfile = sf_open("ofile.wav", SFM_WRITE, &osfinfo);
		p->count = 0;

		p->streams = new hipStream_t[FLIGHT_NUM * 2];
		for (int i = 0; i < 5; i++){
			/*Allocating memory for the inputs*/
			checkCudaErrors(hipMalloc(&(p->d_input[i]), COPY_AMT * sizeof(float)));
			/*Allocating memory for the outputs*/
			checkCudaErrors(hipMalloc(&(p->d_output[i]), FRAMES_PER_BUFFER * HRTF_CHN * sizeof(float)));
			/*Creating the streams*/
			checkCudaErrors(hipStreamCreate(&(p->streams[i * 2])));
			checkCudaErrors(hipStreamCreate(&(p->streams[i * 2 + 1])));
		}
		/*Allocating pinned memory for incoming transfer*/
		checkCudaErrors(hipHostMalloc(&(p->x), COPY_AMT * sizeof(float)));
		/*Allocating pinned memory for outgoing transfer*/
		checkCudaErrors(hipHostMalloc(&(p->intermediate), FRAMES_PER_BUFFER * HRTF_CHN * sizeof(float)));

		/*Setting initial input to 0*/
		for (int i = 0; i < HRTF_LEN - 1; i++){
			p->x[i] = 0.0f;
		}
		p->blockNo = 0;
		hipProfilerStart();

		/*ROUND 1*/
		memcpy(p->x + HRTF_LEN - 1, p->buf + p->count, FRAMES_PER_BUFFER * sizeof(float));
		p->count += FRAMES_PER_BUFFER;
		/*Send B1*/
		checkCudaErrors(hipMemcpyAsync(p->d_input[p->blockNo], p->x, COPY_AMT * sizeof(float), hipMemcpyHostToDevice, p->streams[p->blockNo]));

		/*overlap-save*/
		memcpy(p->x, p->x + FRAMES_PER_BUFFER, (HRTF_LEN - 1) * sizeof(float));
		p->blockNo++;

		/*ROUND 2*/
		/*Copy new input chunk into pinned memory*/
		memcpy(p->x + HRTF_LEN - 1, p->buf + p->count, FRAMES_PER_BUFFER * sizeof(float));
		p->count += FRAMES_PER_BUFFER;
		
		/*Send B2*/
		checkCudaErrors(hipMemcpyAsync(
			p->d_input[p->blockNo], 
			p->x, 
			COPY_AMT * sizeof(float), 
			hipMemcpyHostToDevice, 
			p->streams[p->blockNo * 2])
		);
		/*Process B1*/
		GPUconvolve_hrtf(
			p->d_input[p->blockNo - 1] + HRTF_LEN, 
			p->hrtf_idx, 
			p->d_output[(p->blockNo - 1) % FLIGHT_NUM], 
			FRAMES_PER_BUFFER, 
			p->gain, 
			p->streams+ (p->blockNo - 1) * 2
		);

		/*overlap-save*/
		memcpy(p->x, p->x + FRAMES_PER_BUFFER, (HRTF_LEN - 1) * sizeof(float));
		p->blockNo++;

		/*ROUND 3*/
		/*Copy new input chunk into pinned memory*/
		memcpy(p->x + HRTF_LEN - 1, p->buf + p->count, FRAMES_PER_BUFFER * sizeof(float));
		p->count += FRAMES_PER_BUFFER;

		/*Send B3*/
		fprintf(stderr, "%i %i %i", p->blockNo, p->blockNo - 1, p->blockNo - 2);
		checkCudaErrors(hipMemcpyAsync(
			p->d_input[p->blockNo], 
			p->x, 
			COPY_AMT * sizeof(float), 
			hipMemcpyHostToDevice, 
			p->streams[p->blockNo * 2])
		);
		/*Process B2*/
		GPUconvolve_hrtf(
			p->d_input[p->blockNo - 1] + HRTF_LEN, 
			p->hrtf_idx, 
			p->d_output[(p->blockNo - 1) % FLIGHT_NUM], 
			FRAMES_PER_BUFFER, 
			p->gain, 
			p->streams + (p->blockNo - 1) * 2
		);
		/*Idle B1*/

		/*overlap-save*/
		memcpy(p->x, p->x + FRAMES_PER_BUFFER, (HRTF_LEN - 1) * sizeof(float));
		p->blockNo++;
		
		/*ROUND 4*/
		/*Copy new input chunk into pinned memory*/
		memcpy(p->x + HRTF_LEN - 1, p->buf + p->count, FRAMES_PER_BUFFER * sizeof(float));
		p->count += FRAMES_PER_BUFFER;

		/*Send B4*/
		checkCudaErrors(hipMemcpyAsync(
			p->d_input[p->blockNo % FLIGHT_NUM], 
			p->x, 
			COPY_AMT * sizeof(float), 
			hipMemcpyHostToDevice, 
			p->streams[(p->blockNo) % FLIGHT_NUM * 2])
		);
		/*Process B3*/
		GPUconvolve_hrtf(
			p->d_input[(p->blockNo - 1) % FLIGHT_NUM] + HRTF_LEN, 
			p->hrtf_idx, 
			p->d_output[(p->blockNo - 1) % FLIGHT_NUM], 
			FRAMES_PER_BUFFER, 
			p->gain, 
			p->streams+ (p->blockNo - 1) % FLIGHT_NUM * 2
		);
		/*Idle B2*/

		/*Idle B1*/

		memcpy(p->x, p->x + FRAMES_PER_BUFFER, (HRTF_LEN - 1) * sizeof(float));
		p->blockNo++;

		/*ROUND 5*/
		/*Copy new input chunk into pinned memory*/
		memcpy(p->x + HRTF_LEN - 1, p->buf + p->count, FRAMES_PER_BUFFER * sizeof(float));
		p->count += FRAMES_PER_BUFFER;

		/*Send B5*/
		checkCudaErrors(hipMemcpyAsync(
			p->d_input[p->blockNo % FLIGHT_NUM], 
			p->x, 
			COPY_AMT * sizeof(float), 
			hipMemcpyHostToDevice, 
			p->streams[(p->blockNo) % FLIGHT_NUM * 2])
		);
		/*Process B4*/
		GPUconvolve_hrtf(
			p->d_input[(p->blockNo - 1) % FLIGHT_NUM] + HRTF_LEN, 
			p->hrtf_idx, 
			p->d_output[(p->blockNo - 1) % FLIGHT_NUM], 
			FRAMES_PER_BUFFER, 
			p->gain, 
			p->streams+ (p->blockNo - 1) % FLIGHT_NUM * 2
		);
		/*Idle B3
		/*Idle B2*/

		/*Return B1*/
		checkCudaErrors(hipMemcpyAsync(
			p->intermediate, 
			p->d_output[(p->blockNo - 4) % FLIGHT_NUM], 
			FRAMES_PER_BUFFER * 2 * sizeof(float), 
			hipMemcpyDeviceToHost, 
			p->streams[(p->blockNo - 4) % FLIGHT_NUM * 2])
		);
		memcpy(p->x, p->x + FRAMES_PER_BUFFER, (HRTF_LEN - 1) * sizeof(float));
		p->blockNo++;

		checkCudaErrors(hipDeviceSynchronize());
	#endif
	

#if(DEBUGMODE != 1)
	fprintf(stderr, "\n\n\n\nInitializing PortAudio\n\n\n\n");
	initializePA(SAMPLE_RATE);
	printf("\n\n\n\nStarting playout\n");
	// fprintf(stderr, " %i %i %i %i %i\n", p->blockNo, p->blockNo - 1, p->blockNo - 2, p->blockNo - 3, p->blockNo - 4);
#endif
	///////////////////////////////////////////////////////////////////////////////////////////////////
	/*MAIN FUNCTIONAL LOOP*/
	/*Here to debug without graphics*/
#if DEBUGMODE == 2
	std::this_thread::sleep_for(std::chrono::seconds((p->length)/44100));
	//char merp = getchar();
#else
	graphicsMain(argc, argv, p);
#endif
	
	/*THIS SECTION WILL NOT RUN IF GRAPHICS IS TURNED ON*/
	/*Placed here to properly close files when debugging without graphics*/
	hipProfilerStop();
	
	closeEverything();

	return 0;
}

void closeEverything(){
	closePA();
	sf_close(p->sndfile);
	for(int i = 0; i < 5; i++){
		checkCudaErrors(hipFree(p->d_input[i]));
		checkCudaErrors(hipFree(p->d_output[i]));
		checkCudaErrors(hipStreamSynchronize(p->streams[i * 2]));
		checkCudaErrors(hipStreamSynchronize(p->streams[i * 2 + 1]));
		checkCudaErrors(hipStreamDestroy(p->streams[i * 2]));
		checkCudaErrors(hipStreamDestroy(p->streams[i * 2 + 1]));
	}
	checkCudaErrors(hipHostFree(p->x));
	checkCudaErrors(hipHostFree(p->intermediate));
	free(p->buf);
}