#include "main.cuh"
#include <chrono>
#include <thread>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

void printSize() {
	size_t free = 0, total = 0;
	checkCudaErrors(hipMemGetInfo(&free, &total));
	fprintf(stderr, "GPU Global Memory Stats: Size Free: %.2fMB\tSize Total: %.2fMB\tSize Used: %.2fMB\n", free / 1048576.0f, total / 1048576.0f, (total - free) / 1048576.0f);
}
struct Data data;
struct Data* p = &data;
int main(int argc, char *argv[]){
	//if (argc > 3 ) {
		//fprintf(stderr, "Usage: %s input.wav reverb.wav", argv[0]);
		//return 0;
	//}
	p->num_sources = 1;
	p->all_sources = new SoundSource[p->num_sources]; /*Moving all allocation & initialization into the constructor*/
	printSize();
	#if(DEBUGMODE != 1)
		/*Initialize & read files*/
		cudaFFT(argc, argv, p);
			
		fprintf(stderr, "Opening and Reading HRTF signals\n");
		/*Open & read hrtf files*/

		if (read_hrtf_signals() != 0) {
			exit(EXIT_FAILURE);
		}

		transform_hrtfs();

		fprintf(stderr, "Opening output file\n");
		SF_INFO osfinfo;
		osfinfo.channels = 2;
		osfinfo.samplerate = 44100;
		osfinfo.format = SF_FORMAT_PCM_24 | SF_FORMAT_WAV;
		p->sndfile = sf_open("ofile.wav", SFM_WRITE, &osfinfo);


		printf("Blocks in flight: %i\n", FLIGHT_NUM);
		hipProfilerStart();		
		
		p->blockNo = 0;
		for (int i = 0; i < FLIGHT_NUM; i++) {
			for(int j = 0; j < p->num_sources; j++){
				SoundSource* curr_source = &(p->all_sources[j]);
				/*Copy new input chunk into pinned memory*/
				memcpy(
					curr_source->x[p->blockNo] + (PAD_LEN - FRAMES_PER_BUFFER),  /*Go to the end and work backwards*/
					curr_source->buf + curr_source->count, 
					FRAMES_PER_BUFFER * sizeof(float)
				);
				curr_source->count += FRAMES_PER_BUFFER;

				/*Send*/
				checkCudaErrors(hipMemcpyAsync(
					curr_source->d_input[p->blockNo],
					curr_source->x[p->blockNo],
					PAD_LEN * sizeof(float),
					hipMemcpyHostToDevice,
					curr_source->streams[p->blockNo * STREAMS_PER_FLIGHT])
				);
				if (i == 0) {
					goto end;
				}
				/*Process*/
				//curr_source->fftConvolve(p->blockNo - 1);
				curr_source->interpolateConvolve(p->blockNo - 1);
				if (i == 1) {
					goto end;
				}
				checkCudaErrors(hipMemcpyAsync(
					curr_source->intermediate[(p->blockNo - 2) % FLIGHT_NUM],
					curr_source->d_output[(p->blockNo - 2) % FLIGHT_NUM] + 2 * (PAD_LEN - FRAMES_PER_BUFFER),
					FRAMES_PER_BUFFER * 2 * sizeof(float),
					hipMemcpyDeviceToHost,
					curr_source->streams[(p->blockNo - 2) % FLIGHT_NUM * STREAMS_PER_FLIGHT])
				);
				
			end: /*overlap-save*/
				memmove(
					curr_source->x[(p->blockNo + 1) % FLIGHT_NUM],
					curr_source->x[(p->blockNo) % FLIGHT_NUM] + FRAMES_PER_BUFFER,
					sizeof(float) * (PAD_LEN - FRAMES_PER_BUFFER)
				);
				/*memcpy(
					curr_source->x[(p->blockNo + 1) % FLIGHT_NUM], 
					curr_source->x[p->blockNo] + (PAD_LEN - FRAMES_PER_BUFFER),
					(HRTF_LEN - 1) * sizeof(float)
				);*/
			}
			p->blockNo++;
		}
		checkCudaErrors(hipDeviceSynchronize());
	#endif
	

#if(DEBUGMODE != 1)
	fprintf(stderr, "\n\n\n\nInitializing PortAudio\n\n\n\n");
	initializePA(SAMPLE_RATE);
	printf("\n\n\n\nStarting playout\n");
#endif
	///////////////////////////////////////////////////////////////////////////////////////////////////
	/*MAIN FUNCTIONAL LOOP*/
	/*Here to debug without graphics*/
#if DEBUGMODE == 2
std::this_thread::sleep_for(std::chrono::seconds((p->all_sources[0].length)/44100));
	//char merp = getchar();
#else
	graphicsMain(argc, argv, p);
#endif
	
	/*THIS SECTION WILL NOT RUN IF GRAPHICS IS TURNED ON*/
	/*Placed here to properly close files when debugging without graphics*/
	hipProfilerStop();
	
	closeEverything();

	return 0;
}

void closeEverything(){
	closePA();
	sf_close(p->sndfile);
	for(int source_no = 0; source_no < p->num_sources; source_no++){
		for(int i = 0; i < FLIGHT_NUM; i++){
			checkCudaErrors(hipFree(p->all_sources[source_no].d_input[i]));
			checkCudaErrors(hipFree(p->all_sources[source_no].d_output[i]));
			checkCudaErrors(hipHostFree(p->all_sources[source_no].intermediate[i]));
			checkCudaErrors(hipHostFree(p->all_sources[source_no].x[i]));
			for (int j = 0; j < STREAMS_PER_FLIGHT; j++) {
				checkCudaErrors(hipStreamSynchronize(p->all_sources[source_no].streams[i * STREAMS_PER_FLIGHT + j]));
				checkCudaErrors(hipStreamDestroy(p->all_sources[source_no].streams[i * STREAMS_PER_FLIGHT + j]));
			}			
		}

		free(p->all_sources[source_no].buf);
	}
	
}