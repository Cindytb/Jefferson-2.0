#include "main.cuh"
#include <chrono>
#include <thread>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

void printSize() {
	size_t free = 0, total = 0;
	checkCudaErrors(hipMemGetInfo(&free, &total));
	fprintf(stderr, "GPU Global Memory Stats: Size Free: %.2fMB\tSize Total: %.2fMB\tSize Used: %.2fMB\n", free / 1048576.0f, total / 1048576.0f, (total - free) / 1048576.0f);
}
struct Data data;
struct Data* p = &data;
int main(int argc, char *argv[]){
	//if (argc > 3 ) {
		//fprintf(stderr, "Usage: %s input.wav reverb.wav", argv[0]);
		//return 0;
	//}
	p->num_sources = 1;
	p->all_sources = new SoundSource[p->num_sources]; /*Moving all allocation & initialization into the constructor*/
	printSize();
#if(DEBUGMODE != 1)
		/*Initialize & read files*/
		cudaFFT(argc, argv, p);
			
		fprintf(stderr, "Opening and Reading HRTF signals\n");
		/*Open & read hrtf files*/

		if (read_hrtf_signals() != 0) {
			exit(EXIT_FAILURE);
		}
	#if defined RT_GPU && !defined RT_GPU_TD 
		transform_hrtfs();
	#endif
		fprintf(stderr, "Opening output file\n");
		SF_INFO osfinfo;
		osfinfo.channels = 2;
		osfinfo.samplerate = 44100;
		osfinfo.format = SF_FORMAT_PCM_24 | SF_FORMAT_WAV;
		p->sndfile = sf_open("ofile.wav", SFM_WRITE, &osfinfo);
		
	#ifdef RT_GPU
		printf("Blocks in flight: %i\n", FLIGHT_NUM);
		

		p->blockNo = 0;
		for (int i = 0; i < FLIGHT_NUM; i++) {
			for (int j = 0; j < p->num_sources; j++) {
				SoundSource* curr_source = &(p->all_sources[j]);
				/*Copy new input chunk into pinned memory*/
				int buf_block = p->blockNo;
				memcpy(
					curr_source->x[buf_block] + (PAD_LEN - FRAMES_PER_BUFFER),  /*Go to the end and work backwards*/
					curr_source->buf + curr_source->count,
					FRAMES_PER_BUFFER * sizeof(float)
				);
				curr_source->count += FRAMES_PER_BUFFER;

				curr_source->chunkProcess(buf_block);

				checkCudaErrors(hipDeviceSynchronize());
				/*overlap-save*/
				memmove(
					curr_source->x[(buf_block + 1) % FLIGHT_NUM],
					curr_source->x[buf_block % FLIGHT_NUM] + FRAMES_PER_BUFFER,
					sizeof(float) * (PAD_LEN - FRAMES_PER_BUFFER)
				);
				curr_source->azi += 1;
			}
			p->blockNo++;
			
		}
		checkCudaErrors(hipDeviceSynchronize());
	#endif

#endif
#if(DEBUGMODE % 2 == 0)
	fprintf(stderr, "\n\n\n\nInitializing PortAudio\n\n\n\n");
	initializePA(SAMPLE_RATE);
	printf("\n\n\n\nStarting playout\n");
#endif
	///////////////////////////////////////////////////////////////////////////////////////////////////
	/*MAIN FUNCTIONAL LOOP*/
#if DEBUGMODE == 1
	graphicsMain(argc, argv, p);
#endif
#if DEBUGMODE == 2
	hipProfilerStart();

	int counter = 1;
	while (p->all_sources[0].count < (counter * 44100) % p->all_sources[0].length) {
		std::this_thread::sleep_for(std::chrono::milliseconds(100));
	}
	counter++;
	p->all_sources[0].azi = 2;
	p->all_sources[0].ele = 4;
	p->all_sources[0].updateFromSpherical();

	while (p->all_sources[0].count < (counter * 44100) % p->all_sources[0].length) {
		std::this_thread::sleep_for(std::chrono::milliseconds(100));
	}
	counter++;
	p->all_sources[0].azi = 1;
	p->all_sources[0].ele = 3;
	p->all_sources[0].updateFromSpherical();
	//std::this_thread::sleep_for(std::chrono::seconds(1));
	while (p->all_sources[0].count < (counter * 44100) % p->all_sources[0].length) {
		std::this_thread::sleep_for(std::chrono::milliseconds(100));
	}
	counter++;
	p->all_sources[0].azi = 4;
	p->all_sources[0].ele = 2;
	p->all_sources[0].updateFromSpherical();
	while (p->all_sources[0].count < (counter * 44100) % p->all_sources[0].length) {
		std::this_thread::sleep_for(std::chrono::milliseconds(100));
	}
	counter++;
	p->all_sources[0].azi = 7;
	p->all_sources[0].ele = 9;
	p->all_sources[0].updateFromSpherical();
	while (p->all_sources[0].count < (counter * 44100) % p->all_sources[0].length) {
		std::this_thread::sleep_for(std::chrono::milliseconds(100));
	}
	counter++;
	p->all_sources[0].azi = 0;
	p->all_sources[0].ele = 0;
	p->all_sources[0].updateFromSpherical();
	std::this_thread::sleep_for(std::chrono::seconds(2));
	//char merp = getchar();

	/*THIS SECTION WILL NOT RUN IF GRAPHICS IS TURNED ON*/
	/*Placed here to properly close files when debugging without graphics*/
	hipProfilerStop();

	fprintf(stderr, "Number of function calls: %llu\n", p->all_sources[0].num_calls);
	closeEverything();
#endif
#if DEBUGMODE == 3
	benchmarkTesting();
#endif
	return 0;
}

void closeEverything(){
	closePA();
	checkCudaErrors(hipDeviceSynchronize());
	sf_close(p->sndfile);
	delete[] hrtf;
#ifdef CPU_FD_BASIC
	fftwf_free(fft_hrtf);
#endif
	checkCudaErrors(hipFree(d_hrtf));	
}

void benchmarkTesting(){
	hipProfilerStart();
	float* output = new float[FRAMES_PER_BUFFER * 2];
	int num_iterations = 300;
	for(int i = 0; i < num_iterations; i++){
		callback_func(output, p);
	}
	
	p->all_sources[0].azi = 2;
	p->all_sources[0].ele = 4;
	p->all_sources[0].updateFromSpherical();
	for(int i = 0; i < num_iterations; i++){
		callback_func(output, p);
	}
	p->all_sources[0].azi = 1;
	p->all_sources[0].ele = 3;
	p->all_sources[0].updateFromSpherical();
	for(int i = 0; i < 100; i++){
		callback_func(output, p);
	}
	p->all_sources[0].azi = 4;
	p->all_sources[0].ele = 2;
	p->all_sources[0].updateFromSpherical();
	for(int i = 0; i < num_iterations; i++){
		callback_func(output, p);
	}
	p->all_sources[0].azi = 7;
	p->all_sources[0].ele = 9;
	p->all_sources[0].updateFromSpherical();
	for(int i = 0; i < num_iterations; i++){
		callback_func(output, p);
	}
	p->all_sources[0].azi = 13;
	p->all_sources[0].ele = 14;
	p->all_sources[0].updateFromSpherical();
	for (int i = 0; i < 1000; i++) {
		callback_func(output, p);
	}
}